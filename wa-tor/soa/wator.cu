#include "hip/hip_runtime.h"
#include <chrono>
#include <stdio.h>
#include <assert.h>
#include <inttypes.h>

#include "wa-tor/soa/wator.h"
#include "allocator/allocator_handle.h"

#define SPAWN_THRESHOLD 4
#define ENERGY_BOOST 4
#define ENERGY_START 2

#define GRID_SIZE_X 2048
//#define GRID_SIZE_Y 300

#define OPTION_SHARK_DIE true
#define OPTION_SHARK_SPAWN true
#define OPTION_FISH_SPAWN true

#define THREADS_PER_BLOCK 256
#define NUM_BLOCKS 1024

namespace wa_tor {

__device__ AllocatorT* device_allocator;

// Host side pointer.
AllocatorHandle<AllocatorT>* allocator_handle;


__device__ uint32_t random_number(uint32_t* state, uint32_t max) {
  // Advance and return random state.
  // Source: https://en.wikipedia.org/wiki/Lehmer_random_number_generator
  *state = static_cast<uint32_t>(
      static_cast<uint64_t>(*state) * 1103515245u + 12345) % 2147483648u;
  return ((*state) >> 7) % max;
}

__device__ uint32_t random_number(uint32_t* state) {
  // Advance and return random state.
  // Source: https://en.wikipedia.org/wiki/Lehmer_random_number_generator
  *state = static_cast<uint32_t>(
      static_cast<uint64_t>(*state) * 1103515245u + 12345) % 2147483648u;
  return ((*state) >> 7);
}

__device__ Cell::Cell(uint32_t random_state) : random_state_(random_state),
                                               agent_(nullptr) {
  assert(random_state != 0);
  prepare();
}

__device__ Agent* Cell::agent() const { return agent_; }

__device__ void Cell::decide() {
  if (neighbor_request_[4]) {
    // This cell has priority.
    agent_->set_new_position(this);
  } else {
    uint8_t candidates[4];
    uint8_t num_candidates = 0;

    for (int i = 0; i < 4; ++i) {
      if (neighbor_request_[i]) {
        candidates[num_candidates++] = i;
      }
    }

    if (num_candidates > 0) {
      uint32_t selected_index = random_number(&random_state_, num_candidates);
      neighbors_[candidates[selected_index]]->agent()->set_new_position(this);
    }
  }
}

__device__ void Cell::enter(Agent* agent) {
  assert(agent_ == nullptr);

//#ifndef NDEBUG
//  // Ensure that no two agents are trying to enter this cell at the same time.
//  uint64_t old_val = atomicExch(reinterpret_cast<unsigned long long int*>(&agent_),
//                                reinterpret_cast<unsigned long long int>(agent));
//  assert(old_val == 0);
//#else
  agent_ = agent;
//#endif

  agent->set_position(this);
}

__device__ bool Cell::has_fish() const {
  return agent_ != nullptr && agent_->get_type() == TYPE_ID(AllocatorT, Fish);
}

__device__ bool Cell::has_shark() const {
  return agent_ != nullptr && agent_->get_type() == TYPE_ID(AllocatorT, Shark);
}

__device__ bool Cell::is_free() const { return agent_ == nullptr; }

__device__ void Cell::leave() {
  assert(agent_ != nullptr);
  agent_ = nullptr;
}

__device__ void Cell::prepare() {
  for (int i = 0; i < 5; ++i) { neighbor_request_[i] = false; }
}

__device__ uint32_t* Cell::random_state() { return &random_state_; }

__device__ void Cell::request_random_fish_neighbor() {
  agent_->random_state();
  if (!request_random_neighbor<&Cell::has_fish>(agent_->random_state())) {
    // No fish found. Look for free cell.
    if (!request_random_neighbor<&Cell::is_free>(agent_->random_state())) {
      neighbor_request_[4] = true;
    }
  }
}

__device__ void Cell::request_random_free_neighbor() {
  if (!request_random_neighbor<&Cell::is_free>(agent_->random_state())) {
    neighbor_request_[4] = true;
  }
}

template<bool(Cell::*predicate)() const>
__device__ bool Cell::request_random_neighbor(uint32_t* random_state) {
  uint8_t candidates[4];
  uint8_t num_candidates = 0;

  for (int i = 0; i < 4; ++i) {
    if ((neighbors_[i]->*predicate)()) {
      candidates[num_candidates++] = i;
    }
  }

  if (num_candidates == 0) {
    return false;
  } else {
    uint32_t selected_index = random_number(random_state, num_candidates);
    uint8_t selected = candidates[selected_index];
    uint8_t neighbor_index = (selected + 2) % 4;
    neighbors_[selected]->neighbor_request_[neighbor_index] = true;

    // Check correctness of neighbor calculation.
    assert(neighbors_[selected]->neighbors_[neighbor_index] == this);

    return true;
  }
}

__device__ void Cell::set_neighbors(Cell* left, Cell* top,
                                    Cell* right, Cell* bottom) {
  neighbors_[0] = left;
  neighbors_[1] = top;
  neighbors_[2] = right;
  neighbors_[3] = bottom;
}

__device__ Agent::Agent(uint32_t random_state) : random_state_(random_state) {
  assert(random_state != 0);
}

__device__ uint32_t* Agent::random_state() { return &random_state_; }

__device__ void Agent::set_new_position(Cell* new_pos) {
  // Check for race condition. (This is not bullet proof.)
  assert(new_position_ == position_);

  new_position_ = new_pos;
}

__device__ Cell* Agent::position() const {
  return position_;
}

__device__ void Agent::set_position(Cell* cell) {
  position_ = cell;
}

__device__ Fish::Fish(uint32_t random_state)
    : Agent(random_state), egg_timer_(random_state % SPAWN_THRESHOLD) {
  assert(random_state != 0);
}

__device__ void Fish::prepare() {
  egg_timer_++;
  // Fallback: Stay on current cell.
  new_position_ = position_;

  assert(position_ != nullptr);
  position_->request_random_free_neighbor();
}

__device__ void Fish::update() {
  Cell* old_position = position_;

  if (old_position != new_position_) {
    old_position->leave();
    new_position_->enter(this);

    if (OPTION_FISH_SPAWN && egg_timer_ > SPAWN_THRESHOLD) {
      uint32_t new_random_state = random_number(&random_state_) + 401;
      new_random_state = new_random_state != 0 ? new_random_state
                                               : random_state_;
      auto* new_fish = device_allocator->make_new<Fish>(new_random_state);
      assert(new_fish != nullptr);
      old_position->enter(new_fish);
      egg_timer_ = (uint32_t) 0;
    }
  }
}


__device__ Shark::Shark(uint32_t random_state)
    : Agent(random_state), energy_(ENERGY_START),
      egg_timer_(random_state % SPAWN_THRESHOLD) {
  assert(random_state_ != 0);
}

__device__ void Shark::prepare() {
  egg_timer_++;
  energy_--;

  assert(position_ != nullptr);
  if (OPTION_SHARK_DIE && energy_ == 0) {
    // Do nothing. Shark will die.
  } else {
    // Fallback: Stay on current cell.
    new_position_ = position_;
    position_->request_random_fish_neighbor();
  }
}

__device__ void Shark::update() {
  if (OPTION_SHARK_DIE && energy_ == 0) {
    position_->kill();
  } else {
    Cell* old_position = position_;

    if (old_position != new_position_) {
      if (new_position_->has_fish()) {
        energy_ += ENERGY_BOOST;
        new_position_->kill();
      }

      old_position->leave();
      new_position_->enter(this);

      if (OPTION_SHARK_SPAWN && egg_timer_ > SPAWN_THRESHOLD) {
        assert(random_state_ != 0);
        uint32_t new_random_state = random_number(&random_state_) + 601;
        new_random_state = new_random_state != 0 ? new_random_state
                                                 : random_state_;
        auto* new_shark = device_allocator->make_new<Shark>(new_random_state);
        assert(new_shark != nullptr);
        old_position->enter(new_shark);
        egg_timer_ = 0;
      }
    }
  }
}

__device__ void Cell::kill() {
  assert(agent_ != nullptr);
  device_allocator->free<Agent>(agent_);
  agent_ = nullptr;
}


// ----- KERNELS -----

__device__ Cell* cells[GRID_SIZE_X * GRID_SIZE_Y];

__global__ void create_cells() {
  int tid = threadIdx.x + blockDim.x*blockIdx.x;

  if (tid < GRID_SIZE_Y*GRID_SIZE_X) {
    int x = tid % GRID_SIZE_X;
    int y = tid / GRID_SIZE_X;

    float init_state = __logf(tid + 401);
    uint32_t init_state_int = *reinterpret_cast<uint32_t*>(&init_state);

    // Cell* new_cell = new Cell(init_state_int);
    Cell* new_cell = device_allocator->make_new<Cell>(
        601*x*x*y + init_state_int);
    assert(new_cell != nullptr);
    cells[tid] = new_cell;
  }
}

__global__ void setup_cells() {
  int tid = threadIdx.x + blockDim.x*blockIdx.x;

  if (tid < GRID_SIZE_Y*GRID_SIZE_X) {
    int x = tid % GRID_SIZE_X;
    int y = tid / GRID_SIZE_X;

    Cell* left = x > 0 ? cells[y*GRID_SIZE_X + x - 1]
                       : cells[y*GRID_SIZE_X + GRID_SIZE_X - 1];
    Cell* right = x < GRID_SIZE_X - 1 ? cells[y*GRID_SIZE_X + x + 1]
                                      : cells[y*GRID_SIZE_X];
    Cell* top = y > 0 ? cells[(y - 1)*GRID_SIZE_X + x]
                      : cells[(GRID_SIZE_Y - 1)*GRID_SIZE_X + x];
    Cell* bottom = y < GRID_SIZE_Y - 1 ? cells[(y + 1)*GRID_SIZE_X + x]
                                       : cells[x];

    // left, top, right, bottom
    cells[tid]->set_neighbors(left, top, right, bottom);

    // Initialize with random agent.
    uint32_t agent_type = random_number(cells[tid]->random_state(), 4);
    if (agent_type == 0) {
      auto* agent = device_allocator->make_new<Fish>(
          *(cells[tid]->random_state()));
      assert(agent != nullptr);
      cells[tid]->enter(agent);
    } else if (agent_type == 1) {
      auto* agent = device_allocator->make_new<Shark>(
          *(cells[tid]->random_state()));
      assert(agent != nullptr);
      cells[tid]->enter(agent);
    } else {
      // Free cell.
    }
  }
}

// Problem: It is not easy to keep track of all objects of a class if they are
// dynamically allocated. But we want to benchmark the performance of new/
// delete in CUDA.
// Solution: Fill these arrays in a separate kernel by iterating over all
// cells, storing agents in the respective array slots, and compacting the
// arrays. We do not measure the performance of these steps.
__device__ uint32_t num_sharks = 0;
__device__ Shark* sharks[GRID_SIZE_Y * GRID_SIZE_X];
__device__ uint32_t num_fish = 0;
__device__ Fish*  fish[GRID_SIZE_Y * GRID_SIZE_X];

__global__ void print_checksum() {
  uint64_t chksum = 0;

  // Sorting of the array does not matter in the calculation here.
  for (int i = 0; i < num_sharks; ++i) {
    chksum += *(sharks[i]->position()->random_state()) % 601;
  }

  for (int i = 0; i < num_fish; ++i) {
    chksum += *(fish[i]->position()->random_state()) % 601;
  }

  uint32_t fish_use = device_allocator->DBG_used_slots<Fish>();
  uint32_t fish_num = device_allocator->DBG_allocated_slots<Fish>();
  uint32_t shark_use = device_allocator->DBG_used_slots<Shark>();
  uint32_t shark_num = device_allocator->DBG_allocated_slots<Shark>();

  printf("%" PRIu64, chksum);
  printf(",%u,%u,%u,%u\n",
         fish_use, fish_num, shark_use, shark_num);
}

// One thread per cell.
__global__ void find_fish() {
  int tid = threadIdx.x + blockDim.x*blockIdx.x;

  if (tid < GRID_SIZE_Y*GRID_SIZE_X) {
    if (cells[tid]->has_fish()) {
      uint32_t idx = atomicAdd(&num_fish, 1);
      fish[idx] = reinterpret_cast<Fish*>(cells[tid]->agent());
    }
  }
}

// One thread per cell.
__global__ void find_sharks() {
  int tid = threadIdx.x + blockDim.x*blockIdx.x;

  if (tid < GRID_SIZE_Y*GRID_SIZE_X) {
    if (cells[tid]->has_shark()) {
      uint32_t idx = atomicAdd(&num_sharks, 1);
      sharks[idx] = reinterpret_cast<Shark*>(cells[tid]->agent());
    }
  }
}

__global__ void reset_fish_array() { num_fish = 0; }
__global__ void reset_shark_array() { num_sharks = 0; }

void generate_fish_array() {
  reset_fish_array<<<1, 1>>>();
  gpuErrchk(hipDeviceSynchronize());
  find_fish<<<GRID_SIZE_X*GRID_SIZE_Y/1024 + 1, 1024>>>();
  gpuErrchk(hipDeviceSynchronize());
}

void generate_shark_array() {
  reset_shark_array<<<1, 1>>>();
  gpuErrchk(hipDeviceSynchronize());
  find_sharks<<<GRID_SIZE_X*GRID_SIZE_Y/1024 + 1, 1024>>>();
  gpuErrchk(hipDeviceSynchronize());
}

void step() {
  // --- FISH ---
  allocator_handle->parallel_do<16, Cell, &Cell::prepare>(
      NUM_BLOCKS, THREADS_PER_BLOCK);

  allocator_handle->parallel_do<16, Fish, &Fish::prepare>(
      NUM_BLOCKS, THREADS_PER_BLOCK);

  allocator_handle->parallel_do<16, Cell, &Cell::decide>(
      NUM_BLOCKS, THREADS_PER_BLOCK);

  allocator_handle->parallel_do<16, Fish, &Fish::update>(
      NUM_BLOCKS, THREADS_PER_BLOCK);

  // --- SHARKS ---
  allocator_handle->parallel_do<16, Cell, &Cell::prepare>(
      NUM_BLOCKS, THREADS_PER_BLOCK);

  allocator_handle->parallel_do<16, Shark, &Shark::prepare>(
      NUM_BLOCKS, THREADS_PER_BLOCK);

  allocator_handle->parallel_do<16, Cell, &Cell::decide>(
      NUM_BLOCKS, THREADS_PER_BLOCK);

  allocator_handle->parallel_do<16, Shark, &Shark::update>(
      NUM_BLOCKS, THREADS_PER_BLOCK);
}

void initialize() {
  // Create new allocator.
  allocator_handle = new AllocatorHandle<AllocatorT>();
  AllocatorT* dev_ptr = allocator_handle->device_pointer();
  hipMemcpyToSymbol(HIP_SYMBOL(device_allocator), &dev_ptr, sizeof(AllocatorT*), 0,
                     hipMemcpyHostToDevice);

  create_cells<<<GRID_SIZE_X*GRID_SIZE_Y/1024 + 1, 1024>>>();
  gpuErrchk(hipDeviceSynchronize());
  setup_cells<<<GRID_SIZE_X*GRID_SIZE_Y/1024 + 1, 1024>>>();
  gpuErrchk(hipDeviceSynchronize());
}

__device__ uint32_t d_gui_map[GRID_SIZE_Y * GRID_SIZE_X];
uint32_t gui_map[GRID_SIZE_Y * GRID_SIZE_X];

__global__ void fill_gui_map() {
  int tid = threadIdx.x + blockDim.x*blockIdx.x;

  if (tid < GRID_SIZE_Y*GRID_SIZE_X) {
    if (cells[tid]->agent() != nullptr) {
      d_gui_map[tid] = cells[tid]->agent()->get_type();
    } else {
      d_gui_map[tid] = 0;
    }
  }
}

void update_gui_map() {
  fill_gui_map<<<GRID_SIZE_X*GRID_SIZE_Y/1024 + 1, 1024>>>();
  gpuErrchk(hipDeviceSynchronize());

  hipMemcpyFromSymbol(gui_map, HIP_SYMBOL(d_gui_map), sizeof(uint32_t)*GRID_SIZE_X*GRID_SIZE_Y,
                       0, hipMemcpyDeviceToHost);
  gpuErrchk(hipDeviceSynchronize());
}


void print_stats() {
  generate_fish_array();
  generate_shark_array();
  print_checksum<<<1, 1>>>();
  gpuErrchk(hipDeviceSynchronize());
  printf("           ");
}

int main(int argc, char* arvg[]) {
  AllocatorT::DBG_print_stats();
  
  hipDeviceSetLimit(hipLimitMallocHeapSize, 256*1024*1024);
  size_t heap_size;
  hipDeviceGetLimit(&heap_size, hipLimitMallocHeapSize);
  //printf("CUDA heap size: %lu\n", heap_size);

  initialize();

  int total_time = 0;
  for (int i = 0; i < 500; ++i) {
    auto time_before = std::chrono::system_clock::now();
    step();
    auto time_after = std::chrono::system_clock::now();
    int time_running = std::chrono::duration_cast<std::chrono::microseconds>(
        time_after - time_before).count();
    total_time += time_running;
  }

  printf("%i,%i,", GRID_SIZE_Y, total_time);
  print_stats();
  return 0;
}

}  // namespace wa_tor

int main(int argc, char* arvg[]) {
  return wa_tor::main(0, nullptr);
}
