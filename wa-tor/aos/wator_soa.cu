#include "hip/hip_runtime.h"
#define NDEBUG

#include <chrono>
#include <stdio.h>
#include <assert.h>
#include <inttypes.h>

#include "wa-tor/aos/wator_soa.h"

#define SPAWN_THRESHOLD 4
#define ENERGY_BOOST 4
#define ENERGY_START 2
//#define GRID_SIZE_X 400
//#define GRID_SIZE_Y 300

#define OPTION_SHARK_DIE true
#define OPTION_SHARK_SPAWN true
#define OPTION_FISH_SPAWN true

#define THREADS_PER_BLOCK 256
#define NUM_BLOCKS 1024

namespace wa_tor {

__device__ SoaAllocator<64*64*64*64, Agent, Fish, Shark, Cell> memory_allocator;

template<typename T, typename... Args>
__device__ T* allocate(Args... args) {
  return memory_allocator.make_new<T>(args...);
}

template<typename T>
__device__ void deallocate(T* ptr) {
  memory_allocator.free<T>(ptr);
}

  template<int TypeIndex>
  __device__ void deallocate_untyped(void* ptr) {
    memory_allocator.free_untyped<TypeIndex>(ptr);
  }

__device__ void initialize_allocator() {
  memory_allocator.initialize();
}

__device__ uint32_t random_number(uint32_t* state, uint32_t max) {
  // Advance and return random state.
  // Source: https://en.wikipedia.org/wiki/Lehmer_random_number_generator
  assert(*state != 0);
  *state = static_cast<uint32_t>(
      static_cast<uint64_t>(*state) * 1103515245u + 12345) % 2147483648u;
  return ((*state) >> 7) % max;
}

__device__ uint32_t random_number(uint32_t* state) {
  // Advance and return random state.
  // Source: https://en.wikipedia.org/wiki/Lehmer_random_number_generator
  assert(*state != 0);
  *state = static_cast<uint32_t>(
      static_cast<uint64_t>(*state) * 1103515245u + 12345) % 2147483648u;
  return ((*state) >> 7);
}

__device__ Cell::Cell(uint32_t random_state) : random_state_(random_state),
                                               agent_(nullptr) {
  assert(random_state != 0);
  prepare();
}

__device__ Agent* Cell::agent() const {
  return agent_;
}

__device__ void Cell::decide() {
  // TODO: Not sure why manual type cast is necessary.
  if (arr_neighbor_request(4)) {
    // This cell has priority.
    agent_->set_new_position(this);
  } else {
    uint8_t candidates[4];
    uint8_t num_candidates = 0;

    for (int i = 0; i < 4; ++i) {
      if (arr_neighbor_request(i)) {
        candidates[num_candidates++] = i;
      }
    }

    if (num_candidates > 0) {
      uint32_t selected_index = random_number(&random_state_, num_candidates);
      arr_neighbors(candidates[selected_index])->agent()->set_new_position(this);
    }
  }
}

__device__ void Cell::enter(Agent* agent) {
  assert(agent_ == nullptr);

//#ifndef NDEBUG
//  // Ensure that no two agents are trying to enter this cell at the same time.
//  uint64_t old_val = atomicExch(reinterpret_cast<unsigned long long int*>(&agent_),
//                                reinterpret_cast<unsigned long long int>(agent));
//  assert(old_val == 0);
//#else
  agent_ = agent;
//#endif

  agent->set_position(this);
}

__device__ bool Cell::has_fish() const {
  // TODO: Not sure why typecast is necessary.
  return agent_ != nullptr && ((Agent*)agent_)->type_identifier() == Fish::kTypeId;
}

__device__ bool Cell::has_shark() const {
  // TODO: Not sure why typecast is necessary.
  return agent_ != nullptr && ((Agent*)agent_)->type_identifier() == Shark::kTypeId;
}

__device__ bool Cell::is_free() const {
  return agent_ == nullptr;
}

__device__ void Cell::leave() {
  assert(agent_ != nullptr);
  agent_ = nullptr;
}

__device__ void Cell::prepare() {
  for (int i = 0; i < 5; ++i) {
    arr_neighbor_request(i) = false;
  }
}

__device__ uint32_t* Cell::random_state() {
  return &random_state_;
}

__device__ void Cell::request_random_fish_neighbor() {
  if (!request_random_neighbor<&Cell::has_fish>(agent_->random_state())) {
    // No fish found. Look for free cell.
    if (!request_random_neighbor<&Cell::is_free>(agent_->random_state())) {
      arr_neighbor_request(4) = true;
    }
  }
}

__device__ void Cell::request_random_free_neighbor() {
  if (!request_random_neighbor<&Cell::is_free>(agent_->random_state())) {
    arr_neighbor_request(4) = true;
  }
}

template<bool(Cell::*predicate)() const>
__device__ bool Cell::request_random_neighbor(uint32_t* random_state) {
  uint8_t candidates[4];
  uint8_t num_candidates = 0;

  for (int i = 0; i < 4; ++i) {
    if ((arr_neighbors(i)->*predicate)()) {
      candidates[num_candidates++] = i;
    }
  }

  if (num_candidates == 0) {
    return false;
  } else {
    uint32_t selected_index = random_number(random_state, num_candidates);
    uint8_t selected = candidates[selected_index];
    uint8_t neighbor_index = (selected + 2) % 4;
    arr_neighbors(selected)->arr_neighbor_request(neighbor_index) = true;

    // Check correctness of neighbor calculation.
    assert(arr_neighbors(selected)->arr_neighbors(neighbor_index) == this);

    return true;
  }
}

__device__ void Cell::set_neighbors(Cell* left, Cell* top,
                                    Cell* right, Cell* bottom) {
  arr_neighbors(0) = left;
  arr_neighbors(1) = top;
  arr_neighbors(2) = right;
  arr_neighbors(3) = bottom;
}

__device__ Agent::Agent(uint32_t random_state, uint8_t type_identifier)
    : random_state_(random_state), type_identifier_(type_identifier) {
  assert(random_state != 0);
}

__device__ uint32_t* Agent::random_state() {
  return &random_state_;
}

__device__ void Agent::set_new_position(Cell* new_pos) {
  // Check for race condition. (This is not bullet proof.)
  assert(new_position_ == position_);

  new_position_ = new_pos;
}

__device__ Cell* Agent::position() const {
  return position_;
}

__device__ void Agent::set_position(Cell* cell) {
  position_ = cell;
}

// TODO: Verify that RTTI (dynamic_cast) does not work in device code.
__device__ uint8_t Agent::type_identifier() const {
  return type_identifier_;
}

__device__ Fish::Fish(uint32_t random_state)
    : Agent(random_state, kTypeId), 
      egg_timer_(random_state % SPAWN_THRESHOLD) {
  assert(random_state != 0);
}

__device__ void Fish::prepare() {
  assert(type_identifier() == kTypeId);
  egg_timer_++;
  // Fallback: Stay on current cell.
  new_position_ = position_;

  assert(position_ != nullptr);
  position_->request_random_free_neighbor();
}

__device__ void Fish::update() {
  assert(type_identifier() == kTypeId);
  Cell* old_position = position_;

  if (old_position != new_position_) {
    old_position->leave();
    new_position_->enter(this);

    if (OPTION_FISH_SPAWN && egg_timer_ > SPAWN_THRESHOLD) {
      uint32_t new_random_state = random_number(&random_state_) + 401;
      new_random_state = new_random_state != 0 ? new_random_state
                                               : random_state_;
      auto* new_fish = allocate<Fish>(new_random_state);
      assert(new_fish != nullptr);
      old_position->enter(new_fish);
      egg_timer_ = (uint32_t) 0;
    }
  }
}


__device__ Shark::Shark(uint32_t random_state)
    : Agent(random_state, kTypeId), energy_(ENERGY_START),
      egg_timer_(random_state % SPAWN_THRESHOLD) {
  assert(random_state_ != 0);
}

__device__ void Shark::prepare() {
  assert(type_identifier() == kTypeId);
  egg_timer_++;
  energy_--;

  assert(position_ != nullptr);
  if (OPTION_SHARK_DIE && energy_ == 0) {
    // Do nothing. Shark will die.
  } else {
    // Fallback: Stay on current cell.
    new_position_ = position_;
    position_->request_random_fish_neighbor();
  }
}

__device__ void Shark::update() {
  assert(type_identifier() == kTypeId);

  if (OPTION_SHARK_DIE && energy_ == 0) {
    position_->kill();
  } else {
    Cell* old_position = position_;

    if (old_position != new_position_) {
      if (new_position_->has_fish()) {
        energy_ += ENERGY_BOOST;
        new_position_->kill();
      }

      old_position->leave();
      new_position_->enter(this);

      if (OPTION_SHARK_SPAWN && egg_timer_ > SPAWN_THRESHOLD) {
        assert(random_state_ != 0);
        uint32_t new_random_state = random_number(&random_state_) + 601;
        new_random_state = new_random_state != 0 ? new_random_state
                                                 : random_state_;
        auto* new_shark = allocate<Shark>(new_random_state);
        assert(new_shark != nullptr);
        old_position->enter(new_shark);
        egg_timer_ = 0;
      }
    }
  }
}

__device__ void Cell::kill() {
  assert(agent_ != nullptr);
  if (agent_->type_identifier() == 1) {
    deallocate_untyped<1>(agent_);
  } else if (agent_->type_identifier() == 2) {
    deallocate_untyped<2>(agent_);
  } else {
    // Unknown type.
    assert(false);
  }
  agent_ = nullptr;
}


// ----- KERNELS -----

__device__ Cell* cells[GRID_SIZE_X * GRID_SIZE_Y];

__global__ void create_cells() {
  int tid = threadIdx.x + blockDim.x*blockIdx.x;

  if (tid < GRID_SIZE_Y*GRID_SIZE_X) {
    int x = tid % GRID_SIZE_X;
    int y = tid / GRID_SIZE_X;

    float init_state = __logf(tid + 401);
    uint32_t init_state_int = *reinterpret_cast<uint32_t*>(&init_state);

    // Cell* new_cell = new Cell(init_state_int);
    Cell* new_cell = allocate<Cell>(601*x*x*y + init_state_int);
    assert(new_cell != nullptr);
    cells[tid] = new_cell;
  }
}

__global__ void setup_cells() {
  int tid = threadIdx.x + blockDim.x*blockIdx.x;

  if (tid < GRID_SIZE_Y*GRID_SIZE_X) {
    int x = tid % GRID_SIZE_X;
    int y = tid / GRID_SIZE_X;

    Cell* left = x > 0 ? cells[y*GRID_SIZE_X + x - 1]
                       : cells[y*GRID_SIZE_X + GRID_SIZE_X - 1];
    Cell* right = x < GRID_SIZE_X - 1 ? cells[y*GRID_SIZE_X + x + 1]
                                      : cells[y*GRID_SIZE_X];
    Cell* top = y > 0 ? cells[(y - 1)*GRID_SIZE_X + x]
                      : cells[(GRID_SIZE_Y - 1)*GRID_SIZE_X + x];
    Cell* bottom = y < GRID_SIZE_Y - 1 ? cells[(y + 1)*GRID_SIZE_X + x]
                                       : cells[x];

    // left, top, right, bottom
    cells[tid]->set_neighbors(left, top, right, bottom);

    // Initialize with random agent.
    uint32_t agent_type = random_number(cells[tid]->random_state(), 4);
    if (agent_type == 0) {
      auto* agent = allocate<Fish>(*(cells[tid]->random_state()));
      assert(agent != nullptr);
      cells[tid]->enter(agent);
    } else if (agent_type == 1) {
      auto* agent = allocate<Shark>(*(cells[tid]->random_state()));
      assert(agent != nullptr);
      cells[tid]->enter(agent);
    } else {
      // Free cell.
    }
  }
}

// Problem: It is not easy to keep track of all objects of a class if they are
// dynamically allocated. But we want to benchmark the performance of new/
// delete in CUDA.
// Solution: Fill these arrays in a separate kernel by iterating over all
// cells, storing agents in the respective array slots, and compacting the
// arrays. We do not measure the performance of these steps.
__device__ uint32_t num_sharks = 0;
__device__ Shark* sharks[GRID_SIZE_Y * GRID_SIZE_X];
__device__ uint32_t num_fish = 0;
__device__ Fish*  fish[GRID_SIZE_Y * GRID_SIZE_X];

__global__ void print_checksum() {
  uint64_t chksum = 0;

  // Sorting of the array does not matter in the calculation here.
  for (int i = 0; i < num_sharks; ++i) {
    chksum += *(sharks[i]->position()->random_state()) % 601;
  }

  for (int i = 0; i < num_fish; ++i) {
    chksum += *(fish[i]->position()->random_state()) % 601;
  }

  uint32_t fish_use = memory_allocator.DBG_used_slots<Fish>();
  uint32_t fish_num = memory_allocator.DBG_allocated_slots<Fish>();
  uint32_t shark_use = memory_allocator.DBG_used_slots<Shark>();
  uint32_t shark_num = memory_allocator.DBG_allocated_slots<Shark>();

  printf("%" PRIu64, chksum);
  printf(",%u,%u,%u,%u,%i\n",
         fish_use, fish_num, shark_use, shark_num, (int) GRID_SIZE_X*GRID_SIZE_Y);
}

__global__ void reset_fish_array() {
  num_fish = 0;
}

__global__ void reset_shark_array() {
  num_sharks = 0;
}

// One thread per cell.
__global__ void find_fish() {
  int tid = threadIdx.x + blockDim.x*blockIdx.x;

  if (tid < GRID_SIZE_Y*GRID_SIZE_X) {
    if (cells[tid]->has_fish()) {
      uint32_t idx = atomicAdd(&num_fish, 1);
      fish[idx] = reinterpret_cast<Fish*>(cells[tid]->agent());
    }
  }
}

// One thread per cell.
__global__ void find_sharks() {
  int tid = threadIdx.x + blockDim.x*blockIdx.x;

  if (tid < GRID_SIZE_Y*GRID_SIZE_X) {
    if (cells[tid]->has_shark()) {
      uint32_t idx = atomicAdd(&num_sharks, 1);
      sharks[idx] = reinterpret_cast<Shark*>(cells[tid]->agent());
    }
  }
}

__global__ void find_fish_soa() {
  assert(gridDim.x * blockDim.x == 1);
  num_fish = 0;
  for (int i = 0; i < decltype(memory_allocator)::kN; ++i) {
    if (memory_allocator.is_block_allocated<Fish>(i)) {
      auto* block = memory_allocator.get_block<Fish>(i);
      for (int j = 0; j < Fish::kBlockSize; ++j) {
        if (block->is_slot_allocated(j)) {
          fish[num_fish++] = block->make_pointer(j);
        }
      }
    }
  }
}

__global__ void find_sharks_soa() {
  assert(gridDim.x * blockDim.x == 1);
  num_sharks = 0;
  for (int i = 0; i < decltype(memory_allocator)::kN; ++i) {
    if (memory_allocator.is_block_allocated<Shark>(i)) {
      auto* block = memory_allocator.get_block<Shark>(i);
      for (int j = 0; j < Shark::kBlockSize; ++j) {
        if (block->is_slot_allocated(j)) {
          sharks[num_sharks++] = block->make_pointer(j);
        }
      }
    }
  }
}

__global__ void find_cells_soa() {
  assert(gridDim.x * blockDim.x == 1);
  uint32_t num_cells = 0;
  for (int i = 0; i < decltype(memory_allocator)::kN; ++i) {
    if (memory_allocator.is_block_allocated<Cell>(i)) {
      auto* block = memory_allocator.get_block<Cell>(i);
      for (int j = 0; j < Cell::kBlockSize; ++j) {
        if (block->is_slot_allocated(j)) {
          cells[num_cells++] = block->make_pointer(j);
        }
      }
    }
  }
}

void generate_fish_array_soa() {
  find_fish_soa<<<1,1>>>();
  gpuErrchk(hipDeviceSynchronize());
}

void generate_shark_array_soa() {
  find_sharks_soa<<<1,1>>>();
  gpuErrchk(hipDeviceSynchronize());
}

void generate_fish_array_no_soa() {
  reset_fish_array<<<1, 1>>>();
  gpuErrchk(hipDeviceSynchronize());
  find_fish<<<GRID_SIZE_X*GRID_SIZE_Y/1024 + 1, 1024>>>();
  gpuErrchk(hipDeviceSynchronize());
}

void generate_shark_array_no_soa() {
  reset_shark_array<<<1, 1>>>();
  gpuErrchk(hipDeviceSynchronize());
  find_sharks<<<GRID_SIZE_X*GRID_SIZE_Y/1024 + 1, 1024>>>();
  gpuErrchk(hipDeviceSynchronize());
}

void generate_fish_array() {
  generate_fish_array_soa();
}

void generate_shark_array() {
  generate_shark_array_soa();
}


__global__ void cell_prepare() {
  for (int tid = threadIdx.x + blockDim.x*blockIdx.x;
       tid < GRID_SIZE_Y*GRID_SIZE_X;
       tid += blockDim.x*gridDim.x) {
    cells[tid]->prepare();
  }
}

__global__ void cell_decide() {
  for (int tid = threadIdx.x + blockDim.x*blockIdx.x;
       tid < GRID_SIZE_Y*GRID_SIZE_X;
       tid += blockDim.x*gridDim.x) {
    cells[tid]->decide();
  }
}

__global__ void fish_prepare() {
  for (int tid = threadIdx.x + blockDim.x*blockIdx.x;
       tid < num_fish;
       tid += blockDim.x*gridDim.x) {
    assert(fish[tid] != nullptr);
    fish[tid]->prepare();
  }
}

__global__ void fish_update() {
  for (int tid = threadIdx.x + blockDim.x*blockIdx.x;
       tid < num_fish;
       tid += blockDim.x*gridDim.x) {
    assert(fish[tid] != nullptr);
    fish[tid]->update();
  }
}

__global__ void shark_prepare() {
  for (int tid = threadIdx.x + blockDim.x*blockIdx.x;
       tid < num_sharks;
       tid += blockDim.x*gridDim.x) {
    assert(sharks[tid] != nullptr);
    sharks[tid]->prepare();
  }
}

__global__ void shark_update() {
  for (int tid = threadIdx.x + blockDim.x*blockIdx.x;
       tid < num_sharks;
       tid += blockDim.x*gridDim.x) {
    assert(sharks[tid] != nullptr);
    sharks[tid]->update();
  }
}

void generate_shark_fish_arrays() {
  generate_fish_array();
  generate_shark_array();
}

void step() {
  cell_prepare<<<NUM_BLOCKS, THREADS_PER_BLOCK>>>();
  gpuErrchk(hipDeviceSynchronize());
  fish_prepare<<<NUM_BLOCKS, THREADS_PER_BLOCK>>>();
  gpuErrchk(hipDeviceSynchronize());
  cell_decide<<<NUM_BLOCKS, THREADS_PER_BLOCK>>>();
  gpuErrchk(hipDeviceSynchronize());
  fish_update<<<NUM_BLOCKS, THREADS_PER_BLOCK>>>();
  gpuErrchk(hipDeviceSynchronize());

  cell_prepare<<<NUM_BLOCKS, THREADS_PER_BLOCK>>>();
  gpuErrchk(hipDeviceSynchronize());
  shark_prepare<<<NUM_BLOCKS, THREADS_PER_BLOCK>>>();
  gpuErrchk(hipDeviceSynchronize());
  cell_decide<<<NUM_BLOCKS, THREADS_PER_BLOCK>>>();
  gpuErrchk(hipDeviceSynchronize());
  shark_update<<<NUM_BLOCKS, THREADS_PER_BLOCK>>>();
  gpuErrchk(hipDeviceSynchronize());
}

__global__ void init_memory_system() {
  initialize_allocator();
}

void initialize() {
  init_memory_system<<<GRID_SIZE_X*GRID_SIZE_Y/1024 + 1, 1024>>>();
  gpuErrchk(hipDeviceSynchronize());

  create_cells<<<GRID_SIZE_X*GRID_SIZE_Y/1024 + 1, 1024>>>();
  gpuErrchk(hipDeviceSynchronize());
  setup_cells<<<GRID_SIZE_X*GRID_SIZE_Y/1024 + 1, 1024>>>();
  gpuErrchk(hipDeviceSynchronize());
}

__device__ uint32_t d_gui_map[GRID_SIZE_Y * GRID_SIZE_X];
uint32_t gui_map[GRID_SIZE_Y * GRID_SIZE_X];

__global__ void fill_gui_map() {
  int tid = threadIdx.x + blockDim.x*blockIdx.x;

  if (tid < GRID_SIZE_Y*GRID_SIZE_X) {
    if (cells[tid]->agent() != nullptr) {
      d_gui_map[tid] = cells[tid]->agent()->type_identifier();
    } else {
      d_gui_map[tid] = 0;
    }
  }
}

void update_gui_map() {
  fill_gui_map<<<GRID_SIZE_X*GRID_SIZE_Y/1024 + 1, 1024>>>();
  gpuErrchk(hipDeviceSynchronize());

  hipMemcpyFromSymbol(gui_map, HIP_SYMBOL(d_gui_map), sizeof(uint32_t)*GRID_SIZE_X*GRID_SIZE_Y,
                       0, hipMemcpyDeviceToHost);
  gpuErrchk(hipDeviceSynchronize());
}


int h_num_fish = 0;
int h_num_sharks = 0;


void print_stats() {
  generate_fish_array();
  generate_shark_array();
  //printf("FISH: %i,SHARKS: %i,", h_num_fish, h_num_sharks);
  print_checksum<<<1, 1>>>();
  gpuErrchk(hipDeviceSynchronize());
}

int main(int argc, char* arvg[]) {
  hipDeviceSetLimit(hipLimitMallocHeapSize, 256*1024*1024);

  size_t heap_size;
  hipDeviceGetLimit(&heap_size, hipLimitMallocHeapSize);
  //printf("CUDA heap size: %lu\n", heap_size);

  initialize();

  // To ensure cells are accessed properly (SOA).
  find_cells_soa<<<1, 1>>>();
  gpuErrchk(hipDeviceSynchronize());

  //printf("Computing...");
  int total_time = 0;
  for (int i = 0; i < 100; ++i) {
    if (i%50==0) {
      //render();
    }

    generate_shark_fish_arrays();

    // Printing: RUNNING TIME, NUM_FISH, NUM_SHARKS, CHKSUM, FISH_USE, FISH_ALLOC, SHARK_USE, SHARK_ALLOC
    auto time_before = std::chrono::system_clock::now();
    step();
    auto time_after = std::chrono::system_clock::now();
    int time_running = std::chrono::duration_cast<std::chrono::microseconds>(
        time_after - time_before).count();
    total_time += time_running;
  }
  printf("%i,", total_time);
  print_stats();

  return 0;
}

}  // namespace wa_tor

int main(int argc, char* arvg[]) {
  return wa_tor::main(0, nullptr);
}
