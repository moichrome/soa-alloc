#include "hip/hip_runtime.h"
#include <chrono>
#include <hiprand/hiprand_kernel.h>

#include "dataset.h"
#include "rendering.h"
#include "structure.h"

// Allocator handles.
__device__ AllocatorT* device_allocator;
AllocatorHandle<AllocatorT>* allocator_handle;


__device__ NodeBase::NodeBase(float pos_x, float pos_y)
    : pos_x_(pos_x), pos_y_(pos_y), num_springs_(0) {}


__device__ AnchorNode::AnchorNode(float pos_x, float pos_y)
    : NodeBase(pos_x, pos_y) {}


__device__ AnchorPullNode::AnchorPullNode(float pos_x, float pos_y,
                                          float vel_x, float vel_y)
    : AnchorNode(pos_x, pos_y), vel_x_(vel_x), vel_y_(vel_y) {}


__device__ Node::Node(float pos_x, float pos_y, float mass)
    : NodeBase(pos_x, pos_y), mass_(mass), vel_x_(0.0f), vel_y_(0.0f) {}


__device__ Spring::Spring(NodeBase* p1, NodeBase* p2, float spring_factor,
                          float max_force)
    : p1_(p1), p2_(p2), spring_factor_(spring_factor), force_(0.0f),
      max_force_(max_force), initial_length_(p1->distance_to(p2)) {
  assert(initial_length_ > 0.0f);
  p1_->add_spring(this);
  p2_->add_spring(this);
}


__device__ void NodeBase::add_spring(Spring* spring) {
  int idx = atomicAdd(&num_springs_, 1);
  springs_[idx] = spring;
  assert(idx + 1 <= kMaxDegree);
  assert(spring->p1() == this || spring->p2() == this);
}


__device__ void NodeBase::remove_spring(Spring* spring) {
  // TODO: This won't work if two springs break at the same time.

  int i = 0;
  Spring* s = nullptr;

  do {
    assert(i < kMaxDegree);
    s = springs_[i];
    ++i;
  } while(s != spring);

  for (; i < num_springs_; ++i) {
    springs_[i - 1] = springs_[i];
  }

  --num_springs_;

  if (num_springs_ == 0) {
    device_allocator->free<NodeBase>(this);
  }
}


__device__ float NodeBase::distance_to(NodeBase* other) const {
  float dx = pos_x_ - other->pos_x_;
  float dy = pos_y_ - other->pos_y_;
  float dist_sq = dx*dx + dy*dy;
  return sqrt(dist_sq);
}


__device__ void AnchorPullNode::pull() {
  pos_x_ += vel_x_ * kDt;
  pos_y_ += vel_y_ * kDt;
}


__device__ void Spring::compute_force() {
  float dist = p1_->distance_to(p2_);
  float displacement = max(0.0f, dist - initial_length_);
  force_ = spring_factor_ * displacement;

  if (force_ > max_force_) {
    p1_->remove_spring(this);
    p2_->remove_spring(this);
    device_allocator->free<Spring>(this);
  }
}


__device__ void Node::move() {
  float force_x = 0.0f;
  float force_y = 0.0f;

  for (int i = 0; i < num_springs_; ++i) {
    Spring* s = springs_[i];
    NodeBase* from;
    NodeBase* to;

    if (s->p1() == this) {
      from = this;
      to = s->p2();
    } else {
      assert(s->p2() == this);
      from = this;
      to = s->p1();
    }

    // Calculate unit vector.
    float dx = to->pos_x() - from->pos_x();
    float dy = to->pos_y() - from->pos_y();
    float dist = sqrt(dx*dx + dy*dy);
    float unit_x = dx/dist;
    float unit_y = dy/dist;

    // Apply force.
    force_x += unit_x*s->force();
    force_y += unit_y*s->force();
  }

  // Calculate new velocity and position.
  vel_x_ += force_x*kDt / mass_;
  vel_y_ += force_y*kDt / mass_;
  vel_x_ *= 1.0f - kVelocityDampening;
  vel_y_ *= 1.0f - kVelocityDampening;
  pos_x_ += vel_x_*kDt;
  pos_y_ += vel_y_*kDt;
}


// Only for rendering.
__device__ int dev_num_springs;
__device__ SpringInfo dev_spring_info[kMaxSprings];
int host_num_springs;
SpringInfo host_spring_info[kMaxSprings];

__device__ void Spring::add_to_rendering_array() {
  int idx = atomicAdd(&dev_num_springs, 1);
  dev_spring_info[idx].p1_x = p1_->pos_x();
  dev_spring_info[idx].p1_y = p1_->pos_y();
  dev_spring_info[idx].p2_x = p2_->pos_x();
  dev_spring_info[idx].p2_y = p2_->pos_y();
  dev_spring_info[idx].force = force_;
  dev_spring_info[idx].max_force = max_force_;
}


void transfer_data() {
  int zero = 0;
  hipMemcpyToSymbol(HIP_SYMBOL(dev_num_springs), &zero, sizeof(int), 0,
                     hipMemcpyHostToDevice);
  gpuErrchk(hipDeviceSynchronize());

  allocator_handle->parallel_do<Spring, &Spring::add_to_rendering_array>();
  gpuErrchk(hipDeviceSynchronize());

  hipMemcpyFromSymbol(&host_num_springs, HIP_SYMBOL(dev_num_springs), sizeof(int), 0,
                       hipMemcpyDeviceToHost);
  gpuErrchk(hipDeviceSynchronize());

  hipMemcpyFromSymbol(host_spring_info, HIP_SYMBOL(dev_spring_info),
                       sizeof(SpringInfo)*host_num_springs, 0,
                       hipMemcpyDeviceToHost);
  gpuErrchk(hipDeviceSynchronize());
}


float checksum() {
  transfer_data();
  float result = 0.0f;

  for (int i = 0; i < host_num_springs; ++i) {
    result += host_spring_info[i].p1_x*host_spring_info[i].p2_y
              *host_spring_info[i].force;
  }

  return result;
}


void compute() {
  allocator_handle->parallel_do<Spring, &Spring::compute_force>();
  allocator_handle->parallel_do<Node, &Node::move>();
}


void step() {
  allocator_handle->parallel_do<AnchorPullNode, &AnchorPullNode::pull>();

  for (int i = 0; i < kNumComputeIterations; ++i) {
    compute();
  }

  if (kOptionRender) {
    transfer_data();
    draw(host_num_springs, host_spring_info);
  }
}


__device__ NodeBase* tmp_nodes[kMaxNodes];

__global__ void kernel_create_nodes(DsNode* nodes, int num_nodes) {
  for (int i = threadIdx.x + blockDim.x * blockIdx.x;
       i < num_nodes; i += blockDim.x * gridDim.x) {
    if (nodes[i].type == kTypeNode) {
      tmp_nodes[i] = device_allocator->make_new<Node>(nodes[i].pos_x,
                                                      nodes[i].pos_y,
                                                      nodes[i].mass);
    } else if (nodes[i].type == kTypeAnchorPullNode) {
      tmp_nodes[i] = device_allocator->make_new<AnchorPullNode>(nodes[i].pos_x,
                                                                nodes[i].pos_y,
                                                                nodes[i].vel_x,
                                                                nodes[i].vel_y);
    } else {
      assert(false);
    }
  }
}


__global__ void kernel_create_springs(DsSpring* springs, int num_springs) {
  for (int i = threadIdx.x + blockDim.x * blockIdx.x;
       i < num_springs; i += blockDim.x * gridDim.x) {
    assert(tmp_nodes[springs[i].p1] != nullptr);
    assert(tmp_nodes[springs[i].p2] != nullptr);

    device_allocator->make_new<Spring>(tmp_nodes[springs[i].p1],
                                       tmp_nodes[springs[i].p2],
                                       springs[i].spring_factor,
                                       springs[i].max_force);
  }
}


void load_dataset(Dataset& dataset) {
  DsNode* host_nodes;
  hipMalloc(&host_nodes, sizeof(DsNode)*dataset.nodes.size());
  hipMemcpy(host_nodes, dataset.nodes.data(),
             sizeof(DsNode)*dataset.nodes.size(), hipMemcpyHostToDevice);

  DsSpring* host_springs;
  hipMalloc(&host_springs, sizeof(DsSpring)*dataset.springs.size());
  hipMemcpy(host_springs, dataset.springs.data(),
             sizeof(DsSpring)*dataset.springs.size(), hipMemcpyHostToDevice);
  gpuErrchk(hipDeviceSynchronize());

  kernel_create_nodes<<<128, 128>>>(host_nodes, dataset.nodes.size());
  gpuErrchk(hipDeviceSynchronize());

  kernel_create_springs<<<128, 128>>>(host_springs, dataset.springs.size());
  gpuErrchk(hipDeviceSynchronize());

  hipFree(host_nodes);
  hipFree(host_springs);
}


__global__ void load_example() {
  assert(threadIdx.x == 0 && blockIdx.x == 0);

  float spring_factor = 5.0f;
  float max_force = 100.0f;
  float mass = 500.0f;

  auto* a1 = device_allocator->make_new<AnchorPullNode>(0.1, 0.5, 0.0, -0.02);
  auto* a2 = device_allocator->make_new<AnchorPullNode>(0.3, 0.5, 0.0, -0.02);
  auto* a3 = device_allocator->make_new<AnchorPullNode>(0.5, 0.5, 0.0, -0.02);

  auto* n1 = device_allocator->make_new<Node>(0.05, 0.6, mass);
  auto* n2 = device_allocator->make_new<Node>(0.3, 0.6, mass);
  auto* n3 = device_allocator->make_new<Node>(0.7, 0.6, mass);

  auto* n4 = device_allocator->make_new<Node>(0.2, 0.7, mass);
  auto* n5 = device_allocator->make_new<Node>(0.4, 0.7, mass);
  auto* n6 = device_allocator->make_new<Node>(0.8, 0.7, mass);

  auto* a4 = device_allocator->make_new<AnchorNode>(0.1, 0.9);
  auto* a5 = device_allocator->make_new<AnchorNode>(0.3, 0.9);
  auto* a6 = device_allocator->make_new<AnchorNode>(0.6, 0.9);

  device_allocator->make_new<Spring>(a1, n1, spring_factor, max_force);
  device_allocator->make_new<Spring>(a2, n2, spring_factor, max_force);
  device_allocator->make_new<Spring>(a3, n3, spring_factor, max_force);

  device_allocator->make_new<Spring>(n1, n4, spring_factor, max_force);
  device_allocator->make_new<Spring>(n2, n5, spring_factor, max_force);
  device_allocator->make_new<Spring>(n3, n6, spring_factor, max_force);
  device_allocator->make_new<Spring>(n2, n6, spring_factor, max_force);

  device_allocator->make_new<Spring>(n4, a4, spring_factor, max_force);
  device_allocator->make_new<Spring>(n5, a5, spring_factor, max_force);
  device_allocator->make_new<Spring>(n6, a6, spring_factor, max_force);
}


int main(int /*argc*/, char** /*argv*/) {
  if (kOptionRender) {
    init_renderer();
  }

  // Create new allocator.
  allocator_handle = new AllocatorHandle<AllocatorT>(485000000);
  AllocatorT* dev_ptr = allocator_handle->device_pointer();
  hipMemcpyToSymbol(HIP_SYMBOL(device_allocator), &dev_ptr, sizeof(AllocatorT*), 0,
                     hipMemcpyHostToDevice);

  //load_example<<<1, 1>>>();
  
  Dataset dataset;
  random_dataset(dataset);
  load_dataset(dataset);

  auto time_start = std::chrono::system_clock::now();

  for (int i = 0; i < kNumSteps; ++i) {
    step();
  }

  auto time_end = std::chrono::system_clock::now();
  auto elapsed = time_end - time_start;
  auto millis = std::chrono::duration_cast<std::chrono::milliseconds>(elapsed)
      .count();

  printf("%lu,%lu\n", millis, allocator_handle->DBG_get_enumeration_time());

#ifndef NDEBUG
  printf("Checksum: %f\n", checksum());
#endif  // NDEBUG

  if (kOptionRender) {
    close_renderer();
  }
}
