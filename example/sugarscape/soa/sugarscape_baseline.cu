#include "hip/hip_runtime.h"
#include <assert.h>
#include <chrono>
#include <hiprand/hiprand_kernel.h>
#include <limits>
#include <stdio.h>

#include "configuration.h"
#include "rendering.h"

#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort=true)
{
   if (code != hipSuccess) 
   {
      fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
      if (abort) exit(code);
   }
}

using IndexT = int;

static const int kThreads = 256;
static const int kBlocks = (kSize*kSize + kThreads - 1)/kThreads;
static const IndexT kNullptr = std::numeric_limits<IndexT>::max();

static const char kNoType = 0;
static const char kClassMale = 1;
static const char kClassFemale = 2;

__device__ hiprandState_t* dev_Cell_random_state;
// (No field for agent)
__device__ int* dev_Cell_sugar_diffusion;
__device__ int* dev_Cell_sugar;
__device__ int* dev_Cell_sugar_capacity;
__device__ int* dev_Cell_grow_rate;
// (No field for cell_id)
__device__ char* dev_Cell_Agent_type;
__device__ hiprandState_t* dev_Cell_Agent_random_state;
// (No field for cell)
__device__ IndexT* dev_Cell_Agent_cell_request;
__device__ int* dev_Cell_Agent_vision;
__device__ int* dev_Cell_Agent_age;
__device__ int* dev_Cell_Agent_max_age;
__device__ int* dev_Cell_Agent_sugar;
__device__ int* dev_Cell_Agent_metabolism;
__device__ int* dev_Cell_Agent_endowment;
__device__ bool* dev_Cell_Agent_permission;
__device__ IndexT* dev_Cell_Male_female_request;
__device__ bool* dev_Cell_Male_proposal_accepted;


__device__ float Cell_random_float(IndexT cell_id) {
  return hiprand_uniform(&dev_Cell_random_state[cell_id]);
}


__device__ int Cell_random_int(IndexT cell_id, int a, int b) {
  return hiprand(&dev_Cell_random_state[cell_id]) % (b - a) + a;
}


__device__ float Agent_random_float(IndexT cell_id) {
  return hiprand_uniform(&dev_Cell_Agent_random_state[cell_id]);
}


__device__ bool Cell_is_free(IndexT cell_id) {
  return dev_Cell_Agent_type[cell_id] == kNoType;
}


__device__ void Cell_enter(IndexT cell_id, int agent) {
  assert(cell_id >= 0 && cell_id < kSize*kSize);
  assert(agent >= 0 && agent < kSize*kSize);
  assert(dev_Cell_Agent_type[cell_id] == kNoType);
  assert(dev_Cell_Agent_type[agent] != kNoType);

  dev_Cell_Agent_type[cell_id] = dev_Cell_Agent_type[agent];
  dev_Cell_Agent_random_state[cell_id] = dev_Cell_Agent_random_state[agent];
  dev_Cell_Agent_vision[cell_id] = dev_Cell_Agent_vision[agent];
  dev_Cell_Agent_age[cell_id] = dev_Cell_Agent_age[agent];
  dev_Cell_Agent_max_age[cell_id] = dev_Cell_Agent_max_age[agent];
  dev_Cell_Agent_sugar[cell_id] = dev_Cell_Agent_sugar[agent];
  dev_Cell_Agent_metabolism[cell_id] = dev_Cell_Agent_metabolism[agent];
  dev_Cell_Agent_endowment[cell_id] = dev_Cell_Agent_endowment[agent];
  // No need to copy the other fields.
  dev_Cell_Male_female_request[cell_id] = kNullptr;
  dev_Cell_Male_proposal_accepted[cell_id] = false;
}


__device__ void Cell_leave(IndexT cell_id) {
  assert(dev_Cell_Agent_type[cell_id] != kNoType);
  dev_Cell_Agent_type[cell_id] = kNoType;
  dev_Cell_Agent_permission[cell_id] = false;
  dev_Cell_Agent_cell_request[cell_id] = kNullptr;
  dev_Cell_Male_female_request[cell_id] = kNullptr;
  dev_Cell_Male_proposal_accepted[cell_id] = false;
}


__device__ int xxx69;
__device__ void Agent_harvest_sugar(IndexT cell_id) {
  // Harvest as much sugar as possible.
  // TODO: Do we need two sugar fields here?
  int a = dev_Cell_Agent_sugar[cell_id];
  assert(dev_Cell_Agent_sugar[cell_id] < 9999999);
  //dev_Cell_Agent_sugar[cell_id] += dev_Cell_sugar[cell_id];
  //dev_Cell_sugar[cell_id] = 0;

  int b =dev_Cell_sugar[cell_id];
  assert(b < 9999999);
  xxx69 = b;

  if (a != dev_Cell_Agent_sugar[cell_id]) {
    printf("[%i]   %i -- %i -- %i\n", cell_id, a, dev_Cell_Agent_sugar[cell_id], dev_Cell_sugar[cell_id]);
  }
  //assert(a == dev_Cell_Agent_sugar[cell_id]);
}


__device__ bool Agent_ready_to_mate(IndexT cell_id) {
  // Half of endowment of sugar will go to the child. And the parent still
  // needs some sugar to survive.
  return (dev_Cell_Agent_sugar[cell_id]
          >= dev_Cell_Agent_endowment[cell_id] * 2 / 3)
      && dev_Cell_Agent_age[cell_id] >= kMinMatingAge;
}


__device__ void new_Cell(IndexT cell_id, int seed, int sugar,
                         int sugar_capacity, int max_grow_rate) {
  dev_Cell_sugar[cell_id] = sugar;
  dev_Cell_sugar_capacity[cell_id] = sugar_capacity;
  dev_Cell_Agent_type[cell_id] = kNoType;

  hiprand_init(seed, cell_id, 0, &dev_Cell_random_state[cell_id]);

  // Set random grow rate.
  float r = hiprand_uniform(&dev_Cell_random_state[cell_id]);

  if (r <= 0.01) {
    dev_Cell_grow_rate[cell_id] = max_grow_rate;
  } else if (r <= 0.05) {
    dev_Cell_grow_rate[cell_id] = 0.5*max_grow_rate;
  } else if (r <= 0.07) {
    dev_Cell_grow_rate[cell_id] = 0.25*max_grow_rate;
  } else {
    dev_Cell_grow_rate[cell_id] = 0;
  }
}


__device__ void new_Agent(IndexT cell_id, int vision, int age, int max_age,
                          int endowment, int metabolism) {
  assert(cell_id != kNullptr);
  assert(dev_Cell_Agent_type[cell_id] == kNoType);

  dev_Cell_Agent_cell_request[cell_id] = kNullptr;
  dev_Cell_Agent_vision[cell_id] = vision;
  dev_Cell_Agent_age[cell_id] = age;
  dev_Cell_Agent_max_age[cell_id] = max_age;
  dev_Cell_Agent_sugar[cell_id] = endowment;
  dev_Cell_Agent_endowment[cell_id] = endowment;
  dev_Cell_Agent_metabolism[cell_id] = metabolism;
  dev_Cell_Agent_permission[cell_id] = false;

  hiprand_init(Cell_random_int(cell_id, 0, kSize*kSize), 0, 0,
  	          &dev_Cell_Agent_random_state[cell_id]);
}


__device__ void new_Male(IndexT cell_id, int vision, int age, int max_age,
                         int endowment, int metabolism) {
  new_Agent(cell_id, vision, age, max_age, endowment, metabolism);
  dev_Cell_Male_proposal_accepted[cell_id] = false;
  dev_Cell_Male_female_request[cell_id] = kNullptr;
  dev_Cell_Agent_type[cell_id] = kClassMale;
}


__device__ void new_Female(IndexT cell_id, int vision, int age, int max_age,
                           int endowment, int metabolism) {
  new_Agent(cell_id, vision, age, max_age, endowment, metabolism);
  dev_Cell_Agent_type[cell_id] = kClassFemale;
}


__device__ void Agent_age_and_metabolize(IndexT cell_id) {
  assert(dev_Cell_Agent_type[cell_id] != kNoType);
  bool dead = false;

  dev_Cell_Agent_age[cell_id] = dev_Cell_Agent_age[cell_id] + 1;
  dead = dev_Cell_Agent_age[cell_id] > dev_Cell_Agent_max_age[cell_id];

  dev_Cell_Agent_sugar[cell_id] -= dev_Cell_Agent_metabolism[cell_id];
  dead = dead || (dev_Cell_Agent_sugar[cell_id] <= 0);

  if (dead) {
    Cell_leave(cell_id);
    // No delete in baseline implementation.
  }
}


__device__ void Agent_prepare_move(IndexT cell_id) {
  // Move to cell with the most sugar.
  assert(dev_Cell_Agent_type[cell_id] != kNoType);
  assert(dev_Cell_Agent_cell_request[cell_id] == kNullptr);
  int turn = 0;
  IndexT target_cell = kNullptr;
  int target_sugar = 0;

  IndexT this_x = cell_id % kSize;
  IndexT this_y = cell_id / kSize;

  for (int dx = -dev_Cell_Agent_vision[cell_id];
       dx < dev_Cell_Agent_vision[cell_id] + 1; ++dx) {
    for (int dy = -dev_Cell_Agent_vision[cell_id];
         dy < dev_Cell_Agent_vision[cell_id] + 1; ++dy) {
      IndexT nx = this_x + dx;
      IndexT ny = this_y + dy;
      if ((dx != 0 || dy != 0)
          && nx >= 0 && nx < kSize && ny >= 0 && ny < kSize) {
        IndexT n_id = nx + ny*kSize;

        if (Cell_is_free(n_id)) {
          if (dev_Cell_sugar[n_id] > target_sugar) {
            target_cell = n_id;
            target_sugar = dev_Cell_sugar[n_id];
            turn = 1;
          } else if (dev_Cell_sugar[n_id] == target_sugar) {
            // Select cell with probability 1/turn.
            if (Agent_random_float(cell_id) <= 1.0f/(++turn)) {
              target_cell = n_id;
            }
          }
        }
      }
    }
  }

  dev_Cell_Agent_cell_request[cell_id] = target_cell;
}


__device__ void Agent_update_move(IndexT cell_id) {
  assert(cell_id < kSize*kSize);
  assert(!Cell_is_free(cell_id));
  Agent_harvest_sugar(cell_id);

  int new_cell = cell_id;

  if (dev_Cell_Agent_permission[cell_id]) {
    // Have permission to enter the cell.
    new_cell = dev_Cell_Agent_cell_request[cell_id];
    assert(new_cell != kNullptr);
    assert(new_cell >= 0);
    assert(new_cell < kSize*kSize);
    assert(Cell_is_free(new_cell));
    Cell_enter(new_cell, cell_id);
    Cell_leave(cell_id);
  }

  dev_Cell_Agent_permission[new_cell] = false;
  dev_Cell_Agent_cell_request[new_cell] = kNullptr;
}


__device__ void Cell_prepare_diffuse(IndexT cell_id) {
  dev_Cell_sugar_diffusion[cell_id] =
      kSugarDiffusionRate * dev_Cell_sugar[cell_id];
  int max_diff = kMaxSugarDiffusion;
  if (dev_Cell_sugar_diffusion[cell_id] > max_diff) {
    dev_Cell_sugar_diffusion[cell_id] = max_diff;
  }

  dev_Cell_sugar[cell_id] -= dev_Cell_sugar_diffusion[cell_id];
}


__device__ void Cell_update_diffuse(IndexT cell_id) {
  int new_sugar = 0;
  IndexT this_x = cell_id % kSize;
  IndexT this_y = cell_id / kSize;

  for (int dx = -kMaxVision; dx < kMaxVision + 1; ++dx) {
    for (int dy = -kMaxVision; dy < kMaxVision + 1; ++dy) {
      IndexT nx = this_x + dx;
      IndexT ny = this_y + dy;
        if ((dx != 0 || dy != 0)
            && nx >= 0 && nx < kSize && ny >= 0 && ny < kSize) {
        IndexT n_id = nx + ny*kSize;

        // Add sugar from neighboring 8 cells.
        new_sugar += 0.125f * dev_Cell_sugar_diffusion[n_id];
      }
    }
  }

  dev_Cell_sugar[cell_id] += new_sugar;
}


__device__ void Cell_decide_permission(IndexT cell_id) {
  IndexT selected_agent = kNullptr;
  int turn = 0;
  IndexT this_x = cell_id % kSize;
  IndexT this_y = cell_id / kSize;

  for (int dx = -kMaxVision; dx < kMaxVision + 1; ++dx) {
    for (int dy = -kMaxVision; dy < kMaxVision + 1; ++dy) {
      IndexT nx = this_x + dx;
      IndexT ny = this_y + dy;
      if (nx >= 0 && nx < kSize && ny >= 0 && ny < kSize) {
        IndexT n_id = nx + ny*kSize;

        if (dev_Cell_Agent_type[n_id] != kNoType
            && dev_Cell_Agent_cell_request[n_id] == cell_id) {
          ++turn;

          // Select cell with probability 1/turn.
          if (Cell_random_float(cell_id) <= 1.0f/turn) {
            selected_agent = n_id;
          } else {
            assert(turn > 1);
          }
        }
      }
    }
  }

  assert((turn == 0) == (selected_agent == kNullptr));

  if (selected_agent != kNullptr) {
    assert(Cell_is_free(cell_id));
    assert(dev_Cell_Agent_cell_request[selected_agent] == cell_id);
    dev_Cell_Agent_permission[selected_agent] = true;
  }
}


__device__ void Cell_grow_sugar(IndexT cell_id) {
  dev_Cell_sugar[cell_id] += min(
      dev_Cell_sugar_capacity[cell_id] - dev_Cell_sugar[cell_id],
      dev_Cell_grow_rate[cell_id]);
}


__device__ void Male_propose(IndexT cell_id) {
  if (Agent_ready_to_mate(cell_id)) {
    // Propose to female with highest endowment.
    IndexT target_agent = kNullptr;
    int target_sugar = -1;

    IndexT this_x = cell_id % kSize;
    IndexT this_y = cell_id / kSize;

    for (int dx = -dev_Cell_Agent_vision[cell_id];
         dx < dev_Cell_Agent_vision[cell_id] + 1; ++dx) {
      for (int dy = -dev_Cell_Agent_vision[cell_id];
           dy < dev_Cell_Agent_vision[cell_id] + 1; ++dy) {
        IndexT nx = this_x + dx;
        IndexT ny = this_y + dy;
        if (nx >= 0 && nx < kSize && ny >= 0 && ny < kSize) {
          IndexT n_id = nx + ny*kSize;

          if (dev_Cell_Agent_type[n_id] == kClassFemale
              && Agent_ready_to_mate(n_id)) {
            if (dev_Cell_Agent_sugar[n_id] > target_sugar) {
              target_agent = n_id;
              target_sugar = dev_Cell_Agent_sugar[n_id];
            }
          }
        }
      }
    }

    assert((target_sugar == -1) == (target_agent == kNullptr));
    dev_Cell_Male_female_request[cell_id] = target_agent;
  }
}


__device__ void Male_propose_offspring_target(IndexT cell_id) {
  if (dev_Cell_Male_proposal_accepted[cell_id]) {
    assert(dev_Cell_Male_female_request[cell_id] != kNullptr);

    // Select a random cell.
    IndexT target_cell = kNullptr;
    int turn = 0;

    IndexT this_x = cell_id % kSize;
    IndexT this_y = cell_id / kSize;

    for (int dx = -dev_Cell_Agent_vision[cell_id];
         dx < dev_Cell_Agent_vision[cell_id] + 1; ++dx) {
      for (int dy = -dev_Cell_Agent_vision[cell_id];
           dy < dev_Cell_Agent_vision[cell_id] + 1; ++dy) {
        IndexT nx = this_x + dx;
        IndexT ny = this_y + dy;
        if ((dx != 0 || dy != 0)
            && nx >= 0 && nx < kSize && ny >= 0 && ny < kSize) {
          IndexT n_id = nx + ny*kSize;

          if (Cell_is_free(n_id)) {
            ++turn;

            // Select cell with probability 1/turn.
            if (Agent_random_float(cell_id) <= 1.0f/turn) {
              target_cell = n_id;
            }
          }
        }
      }
    }

    assert((turn == 0) == (target_cell == kNullptr));
    dev_Cell_Agent_cell_request[cell_id] = target_cell;
  }
}


__device__ void Male_mate(IndexT cell_id) {
  if (dev_Cell_Male_proposal_accepted[cell_id]
      && dev_Cell_Agent_permission[cell_id]) {
    assert(dev_Cell_Male_female_request[cell_id] != kNullptr);
    assert(dev_Cell_Agent_cell_request[cell_id] != kNullptr);

    // Take sugar from endowment.
    int c_endowment = (dev_Cell_Agent_endowment[cell_id]
        + dev_Cell_Agent_endowment[dev_Cell_Male_female_request[cell_id]]) / 2;
    dev_Cell_Agent_sugar[cell_id] -= dev_Cell_Agent_endowment[cell_id] / 2;
    dev_Cell_Agent_sugar[dev_Cell_Male_female_request[cell_id]]
        -= dev_Cell_Agent_endowment[dev_Cell_Male_female_request[cell_id]] / 2;

    // Calculate other properties.
    int c_vision = (dev_Cell_Agent_vision[cell_id]
        + dev_Cell_Agent_vision[dev_Cell_Male_female_request[cell_id]]) / 2;
    int c_max_age = (dev_Cell_Agent_max_age[cell_id]
        + dev_Cell_Agent_max_age[dev_Cell_Male_female_request[cell_id]]) / 2;
    int c_metabolism = (dev_Cell_Agent_metabolism[cell_id]
        + dev_Cell_Agent_metabolism[dev_Cell_Male_female_request[cell_id]]) / 2;


    // Create agent.
    assert(dev_Cell_Agent_cell_request[cell_id] != kNullptr);
    assert(dev_Cell_Agent_type[dev_Cell_Agent_cell_request[cell_id]] == kNoType);

    if (Agent_random_float(cell_id) <= 0.5f) {
      new_Male(dev_Cell_Agent_cell_request[cell_id],
               2*c_vision, /*age=*/ 0, c_max_age, c_endowment, c_metabolism);
    } else {
      new_Female(dev_Cell_Agent_cell_request[cell_id],
                 c_vision, /*age=*/ 0, c_max_age, c_endowment, c_metabolism);
    }
  }

  dev_Cell_Agent_permission[cell_id] = false;
  dev_Cell_Male_proposal_accepted[cell_id] = false;
  dev_Cell_Male_female_request[cell_id] = kNullptr;
  dev_Cell_Agent_cell_request[cell_id] = kNullptr;
}


__device__ void Female_decide_proposal(IndexT cell_id) {
  IndexT selected_agent = kNullptr;
  int selected_sugar = -1;
  IndexT this_x = cell_id % kSize;
  IndexT this_y = cell_id / kSize;

  for (int dx = -kMaxVision; dx < kMaxVision + 1; ++dx) {
    for (int dy = -kMaxVision; dy < kMaxVision + 1; ++dy) {
      IndexT nx = this_x + dx;
      IndexT ny = this_y + dy;
      if (nx >= 0 && nx < kSize && ny >= 0 && ny < kSize) {
        IndexT n_id = nx + ny*kSize;

        if (dev_Cell_Agent_type[n_id] == kClassMale) {
          if (dev_Cell_Male_female_request[n_id] == cell_id
              && dev_Cell_Agent_sugar[n_id] > selected_sugar) {
            selected_agent = n_id;
            selected_sugar = dev_Cell_Agent_sugar[n_id];
          }
        }
      }
    }
  }

  assert((selected_sugar == -1) == (selected_agent == kNullptr));

  if (selected_agent != kNullptr) {
    dev_Cell_Male_proposal_accepted[selected_agent] = true;
  }
}


__global__ void kernel_Cell_grow_sugar() {
  for (IndexT i = threadIdx.x + blockIdx.x * blockDim.x;
       i < kSize*kSize; i += blockDim.x * gridDim.x) {
    Cell_grow_sugar(i);
  }
}


__global__ void kernel_Cell_prepare_diffuse() {
  for (IndexT i = threadIdx.x + blockIdx.x * blockDim.x;
       i < kSize*kSize; i += blockDim.x * gridDim.x) {
    Cell_prepare_diffuse(i);
  }
}


__global__ void kernel_Cell_update_diffuse() {
  for (IndexT i = threadIdx.x + blockIdx.x * blockDim.x;
       i < kSize*kSize; i += blockDim.x * gridDim.x) {
    Cell_update_diffuse(i);
  }
}


__global__ void kernel_Agent_age_and_metabolize() {
  for (IndexT i = threadIdx.x + blockIdx.x * blockDim.x;
       i < kSize*kSize; i += blockDim.x * gridDim.x) {
    if (dev_Cell_Agent_type[i] != kNoType) {
      Agent_age_and_metabolize(i);
    }
  }
}


__global__ void kernel_Agent_prepare_move() {
  for (IndexT i = threadIdx.x + blockIdx.x * blockDim.x;
       i < kSize*kSize; i += blockDim.x * gridDim.x) {
    if (dev_Cell_Agent_type[i] != kNoType) {
      Agent_prepare_move(i);
    }
  }
}


__global__ void kernel_Cell_decide_permission() {
  for (IndexT i = threadIdx.x + blockIdx.x * blockDim.x;
       i < kSize*kSize; i += blockDim.x * gridDim.x) {
    Cell_decide_permission(i);
  }
}


__global__ void kernel_Agent_update_move() {
  for (IndexT i = threadIdx.x + blockIdx.x * blockDim.x;
       i < kSize*kSize; i += blockDim.x * gridDim.x) {
    if (dev_Cell_Agent_type[i] != kNoType) {
      Agent_update_move(i);
    }
  }
}


__global__ void kernel_Male_propose() {
  for (IndexT i = threadIdx.x + blockIdx.x * blockDim.x;
       i < kSize*kSize; i += blockDim.x * gridDim.x) {
    if (dev_Cell_Agent_type[i] == kClassMale) {
      Male_propose(i);
    }
  }
}


__global__ void kernel_Female_decide_proposal() {
  for (IndexT i = threadIdx.x + blockIdx.x * blockDim.x;
       i < kSize*kSize; i += blockDim.x * gridDim.x) {
    if (dev_Cell_Agent_type[i] == kClassFemale) {
      Female_decide_proposal(i);
    }
  }
}


__global__ void kernel_Male_propose_offspring_target() {
  for (IndexT i = threadIdx.x + blockIdx.x * blockDim.x;
       i < kSize*kSize; i += blockDim.x * gridDim.x) {
    if (dev_Cell_Agent_type[i] == kClassMale) {
      Male_propose_offspring_target(i);
    }
  }
}


__global__ void kernel_Male_mate() {
  for (IndexT i = threadIdx.x + blockIdx.x * blockDim.x;
       i < kSize*kSize; i += blockDim.x * gridDim.x) {
    if (dev_Cell_Agent_type[i] == kClassMale) {
      Male_mate(i);
    }
  }
}


void step() {
  /*
  kernel_Cell_grow_sugar<<<kBlocks, kThreads>>>();
  gpuErrchk(hipDeviceSynchronize());

  kernel_Cell_prepare_diffuse<<<kBlocks, kThreads>>>();
  gpuErrchk(hipDeviceSynchronize());

  kernel_Cell_update_diffuse<<<kBlocks, kThreads>>>();
  gpuErrchk(hipDeviceSynchronize());
  */

  kernel_Agent_age_and_metabolize<<<kBlocks, kThreads>>>();
  gpuErrchk(hipDeviceSynchronize());

  kernel_Agent_prepare_move<<<kBlocks, kThreads>>>();
  gpuErrchk(hipDeviceSynchronize());

  kernel_Cell_decide_permission<<<kBlocks, kThreads>>>();
  gpuErrchk(hipDeviceSynchronize());

  kernel_Agent_update_move<<<kBlocks, kThreads>>>();
  gpuErrchk(hipDeviceSynchronize());

/*
  kernel_Male_propose<<<kBlocks, kThreads>>>();
  gpuErrchk(hipDeviceSynchronize());

  kernel_Female_decide_proposal<<<kBlocks, kThreads>>>();
  gpuErrchk(hipDeviceSynchronize());

  kernel_Male_propose_offspring_target<<<kBlocks, kThreads>>>();
  gpuErrchk(hipDeviceSynchronize());

  kernel_Cell_decide_permission<<<kBlocks, kThreads>>>();
  gpuErrchk(hipDeviceSynchronize());

  kernel_Male_mate<<<kBlocks, kThreads>>>();
  gpuErrchk(hipDeviceSynchronize());
*/
}


__global__ void create_cells() {
  for (IndexT i = threadIdx.x + blockDim.x * blockIdx.x;
       i < kSize*kSize; i += blockDim.x * gridDim.x) {
    new_Cell(i, kSeed, /*sugar=*/ 0, kSugarCapacity, /*max_grow_rate=*/ 50);
  }
}


__global__ void create_agents() {
  for (IndexT i = threadIdx.x + blockDim.x * blockIdx.x;
       i < kSize*kSize; i += blockDim.x * gridDim.x) {
    float r = Cell_random_float(i);
    int c_vision = kMaxVision/2 + Cell_random_int(i, 0, kMaxVision/2);
    int c_max_age = kMaxAge*2/3 + Cell_random_int(i, 0, kMaxAge/3);
    int c_endowment = kMaxEndowment/4
                      + Cell_random_int(i, 0, kMaxEndowment*3/4);
    int c_metabolism = kMaxMetabolism/3
                       + Cell_random_int(i, 0, kMaxMetabolism*2/3);

    if (r < kProbMale) {
      // Create male agent.
      new_Male(i, 2*c_vision, /*age=*/ 0, c_max_age, c_endowment, c_metabolism);
    } else if (r < kProbMale + kProbFemale) {
      // Create female agent.
      new_Female(i, c_vision, /*age=*/ 0, c_max_age, c_endowment, c_metabolism);
    }   // else: Do not create agent.
  }
}


void initialize_simulation() {
  create_cells<<<128, 128>>>();
  gpuErrchk(hipDeviceSynchronize());

  create_agents<<<128, 128>>>();
  gpuErrchk(hipDeviceSynchronize());
}


int data_Cell_sugar[kSize*kSize];
char data_Cell_type[kSize*kSize];
CellInfo host_cell_info[kSize*kSize];

void transfer_data(int* host_Cell_sugar, char* host_Cell_Agent_type) {
  hipMemcpy(data_Cell_sugar, host_Cell_sugar, sizeof(int)*kSize*kSize,
             hipMemcpyDeviceToHost);
  hipMemcpy(data_Cell_type, host_Cell_Agent_type, sizeof(char)*kSize*kSize,
             hipMemcpyDeviceToHost);
  gpuErrchk(hipDeviceSynchronize());

  for (int i = 0; i < kSize*kSize; ++i) {
    host_cell_info[i].sugar = data_Cell_sugar[i];
    host_cell_info[i].agent_type = data_Cell_type[i];
  }
}


int checksum(int* host_Cell_sugar, char* host_Cell_Agent_type) {
  transfer_data(host_Cell_sugar, host_Cell_Agent_type);
  int result = 0;
  for (int i = 0; i < kSize*kSize; ++i) {
    result += data_Cell_type[i];
    //result %= 12456789;
  }
  return result;
}


int main(int /*argc*/, char** /*argv*/) {
  if (kOptionRender) {
    init_renderer();
  }

  // Allocate memory.
  hiprandState_t* host_Cell_random_state;
  hipMalloc(&host_Cell_random_state, sizeof(hiprandState_t)*kSize*kSize);
  hipMemcpyToSymbol(HIP_SYMBOL(dev_Cell_random_state), &host_Cell_random_state,
                     sizeof(hiprandState_t*), 0, hipMemcpyHostToDevice);

  int* host_Cell_sugar_diffusion;
  hipMalloc(&host_Cell_sugar_diffusion, sizeof(int)*kSize*kSize);
  hipMemcpyToSymbol(HIP_SYMBOL(dev_Cell_sugar_diffusion), &host_Cell_sugar_diffusion,
                     sizeof(int*), 0, hipMemcpyHostToDevice);

  int* host_Cell_sugar;
  hipMalloc(&host_Cell_sugar, sizeof(int)*kSize*kSize);
  hipMemcpyToSymbol(HIP_SYMBOL(dev_Cell_sugar), &host_Cell_sugar,
                     sizeof(int*), 0, hipMemcpyHostToDevice);

  int* host_Cell_sugar_capacity;
  hipMalloc(&host_Cell_sugar_capacity, sizeof(int)*kSize*kSize);
  hipMemcpyToSymbol(HIP_SYMBOL(dev_Cell_sugar_capacity), &host_Cell_sugar_capacity,
                     sizeof(int*), 0, hipMemcpyHostToDevice);

  int* host_Cell_grow_rate;
  hipMalloc(&host_Cell_grow_rate, sizeof(int)*kSize*kSize);
  hipMemcpyToSymbol(HIP_SYMBOL(dev_Cell_grow_rate), &host_Cell_grow_rate,
                     sizeof(int*), 0, hipMemcpyHostToDevice);

  char* host_Cell_Agent_type;
  hipMalloc(&host_Cell_Agent_type, sizeof(char)*kSize*kSize);
  hipMemcpyToSymbol(HIP_SYMBOL(dev_Cell_Agent_type), &host_Cell_Agent_type,
                     sizeof(char*), 0, hipMemcpyHostToDevice);

  hiprandState_t* host_Cell_Agent_random_state;
  hipMalloc(&host_Cell_Agent_random_state, sizeof(hiprandState_t)*kSize*kSize);
  hipMemcpyToSymbol(HIP_SYMBOL(dev_Cell_Agent_random_state), &host_Cell_Agent_random_state,
                     sizeof(hiprandState_t*), 0, hipMemcpyHostToDevice);

  IndexT* host_Cell_Agent_cell_request;
  hipMalloc(&host_Cell_Agent_cell_request, sizeof(IndexT)*kSize*kSize);
  hipMemcpyToSymbol(HIP_SYMBOL(dev_Cell_Agent_cell_request), &host_Cell_Agent_cell_request,
                     sizeof(IndexT*), 0, hipMemcpyHostToDevice);

  int* host_Cell_Agent_vision;
  hipMalloc(&host_Cell_Agent_vision, sizeof(int)*kSize*kSize);
  hipMemcpyToSymbol(HIP_SYMBOL(dev_Cell_Agent_vision), &host_Cell_Agent_vision,
                     sizeof(int*), 0, hipMemcpyHostToDevice);

  int* host_Cell_Agent_age;
  hipMalloc(&host_Cell_Agent_age, sizeof(int)*kSize*kSize);
  hipMemcpyToSymbol(HIP_SYMBOL(dev_Cell_Agent_age), &host_Cell_Agent_age,
                     sizeof(int*), 0, hipMemcpyHostToDevice);

  int* host_Cell_Agent_max_age;
  hipMalloc(&host_Cell_Agent_max_age, sizeof(int)*kSize*kSize);
  hipMemcpyToSymbol(HIP_SYMBOL(dev_Cell_Agent_max_age), &host_Cell_Agent_max_age,
                     sizeof(int*), 0, hipMemcpyHostToDevice);

  int* host_Cell_Agent_sugar;
  hipMalloc(&host_Cell_Agent_sugar, sizeof(int)*kSize*kSize);
  hipMemcpyToSymbol(HIP_SYMBOL(dev_Cell_Agent_sugar), &host_Cell_Agent_sugar,
                     sizeof(int*), 0, hipMemcpyHostToDevice);

  int* host_Cell_Agent_metabolism;
  hipMalloc(&host_Cell_Agent_metabolism, sizeof(int)*kSize*kSize);
  hipMemcpyToSymbol(HIP_SYMBOL(dev_Cell_Agent_metabolism), &host_Cell_Agent_metabolism,
                     sizeof(int*), 0, hipMemcpyHostToDevice);

  int* host_Cell_Agent_endowment;
  hipMalloc(&host_Cell_Agent_endowment, sizeof(int)*kSize*kSize);
  hipMemcpyToSymbol(HIP_SYMBOL(dev_Cell_Agent_endowment), &host_Cell_Agent_endowment,
                     sizeof(int*), 0, hipMemcpyHostToDevice);

  bool* host_Cell_Agent_permission;
  hipMalloc(&host_Cell_Agent_permission, sizeof(bool)*kSize*kSize);
  hipMemcpyToSymbol(HIP_SYMBOL(dev_Cell_Agent_permission), &host_Cell_Agent_permission,
                     sizeof(bool*), 0, hipMemcpyHostToDevice);

  IndexT* host_Cell_Male_female_request;
  hipMalloc(&host_Cell_Male_female_request, sizeof(IndexT)*kSize*kSize);
  hipMemcpyToSymbol(HIP_SYMBOL(dev_Cell_Male_female_request),
                     &host_Cell_Male_female_request,
                     sizeof(IndexT*), 0, hipMemcpyHostToDevice);

  bool* host_Cell_Male_proposal_accepted;
  hipMalloc(&host_Cell_Male_proposal_accepted, sizeof(bool)*kSize*kSize);
  hipMemcpyToSymbol(HIP_SYMBOL(dev_Cell_Male_proposal_accepted),
                     &host_Cell_Male_proposal_accepted,
                     sizeof(bool*), 0, hipMemcpyHostToDevice);
  assert(host_Cell_Male_proposal_accepted != nullptr);

  gpuErrchk(hipDeviceSynchronize());

  initialize_simulation();

  auto time_start = std::chrono::system_clock::now();

  for (int i = 0; i < kNumIterations; ++i) {
    //printf("%i\n", i);
    //printf("Checksum: %i\n", checksum(host_Cell_sugar, host_Cell_Agent_type));
    step();

    if (kOptionRender) {
      transfer_data(host_Cell_sugar, host_Cell_Agent_type);
      draw(host_cell_info);
    }
  }

  auto time_end = std::chrono::system_clock::now();
  auto elapsed = time_end - time_start;
  auto millis = std::chrono::duration_cast<std::chrono::milliseconds>(elapsed)
      .count();

  printf("Time: %lu ms\n", millis);

  if (kOptionRender) {
    close_renderer();
  }

  printf("Checksum: %i\n", checksum(host_Cell_sugar, host_Cell_Agent_type));
  return 0;

  // TODO: Free CUDA memory.
}
