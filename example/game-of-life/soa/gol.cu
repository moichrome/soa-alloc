#include "hip/hip_runtime.h"
#include <chrono>

#include "example/game-of-life/soa/gol.h"
#include "example/game-of-life/soa/configuration.h"
#include "example/game-of-life/soa/dataset_loader.h"
#include "example/game-of-life/soa/rendering.h"


// Allocator handles.
AllocatorHandle<AllocatorT>* allocator_handle;
__device__ AllocatorT* device_allocator;


// Rendering array.
// TODO: Fix variable names.
__device__ char* device_render_cells;
char* host_render_cells;
char* d_device_render_cells;


// Dataset.
__device__ int SIZE_X;
__device__ int SIZE_Y;
__device__ Cell** cells;
dataset_t dataset;


__device__ Cell::Cell() : agent_(nullptr) {}


__device__ Agent* Cell::agent() { return agent_; }


__device__ bool Cell::is_empty() { return agent_ == nullptr; }


__device__ bool Cell::is_alive() {
  return !is_empty() && agent_->get_type() == TYPE_ID(AllocatorT, Alive);
}


__device__ bool Cell::is_candidate() {
  return !is_empty() && agent_->get_type() == TYPE_ID(AllocatorT, Candidate);
}


__device__ Agent::Agent(int cell_id)
    : cell_id_(cell_id), action_(kActionNone) {}


__device__ int Agent::cell_id() { return cell_id_; }


__device__ int Agent::num_alive_neighbors() {
  int cell_x = cell_id_ % SIZE_X;
  int cell_y = cell_id_ / SIZE_X;
  int result = 0;

  for (int dx = -1; dx < 2; ++dx) {
    for (int dy = -1; dy < 2; ++dy) {
      int nx = cell_x + dx;
      int ny = cell_y + dy;

      if (nx > -1 && nx < SIZE_X && ny > -1 && ny < SIZE_Y) {
        if (cells[ny*SIZE_X + nx]->is_alive()) {
          result++;
        }
      }
    }
  }

  return result;
}


__device__ Alive::Alive(int cell_id) : Agent(cell_id), is_new_(true) {}


__device__ void Alive::prepare() {
  is_new_ = false;

  // Also counts this object itself.
  int alive_neighbors = num_alive_neighbors() - 1;

  if (alive_neighbors < 2 || alive_neighbors > 3) {
    action_ = kActionDie;
  }
}


__device__ void Alive::update() {
  int cid = cell_id_;

  // TODO: Consider splitting in two classes for less divergence.
  if (is_new_) {
    // Create candidates in neighborhood.
    create_candidates();
  } else {
    if (action_ == kActionDie) {
      // Replace with Candidate. Or should we?
      cells[cid]->agent_ =
          device_allocator->make_new<Candidate>(cid);
      device_allocator->free<Alive>(this);
    }
  }
}


__device__ void Alive::create_candidates() {
  assert(is_new_);

  // TODO: Consolidate with Agent::num_alive_neighbors().
  int cell_x = cell_id_ % SIZE_X;
  int cell_y = cell_id_ / SIZE_X;

  for (int dx = -1; dx < 2; ++dx) {
    for (int dy = -1; dy < 2; ++dy) {
      int nx = cell_x + dx;
      int ny = cell_y + dy;

      if (nx > -1 && nx < SIZE_X && ny > -1 && ny < SIZE_Y) {
        if (cells[ny*SIZE_X + nx]->is_empty()) {
          // Candidate should be created here.
          //maybe_create_candidate(nx, ny);
          Agent** agent_addr = cells[ny*SIZE_X + nx]->agent_.data_ptr();

          if (atomicCAS(reinterpret_cast<unsigned long long int*>(agent_addr),
                        /*compare=*/ 0ULL,
                        /*val=*/ 1ULL) == 0ULL) {
            cells[ny*SIZE_X + nx]->agent_ = device_allocator->make_new<Candidate>(ny*SIZE_X + nx);
          }
        }
      }
    }
  }
}


__device__ void Alive::maybe_create_candidate(int x, int y) {
  // Check neighborhood of cell to determine who should create Candidate.
  for (int dx = -1; dx < 2; ++dx) {
    for (int dy = -1; dy < 2; ++dy) {
      int nx = x + dx;
      int ny = y + dy;

      if (nx > -1 && nx < SIZE_X && ny > -1 && ny < SIZE_Y) {
        if (cells[ny*SIZE_X + nx]->is_alive()) {
          Alive* alive = static_cast<Alive*>(cells[ny*SIZE_X + nx]->agent());
          if (alive->is_new_) {
            if (alive == this) {
              // Create candidate now.
              cells[y*SIZE_X + x]->agent_ =
                  device_allocator->make_new<Candidate>(y*SIZE_X + x);
            }  // else: Created by other thread.

            return;
          }
        }
      }
    }
  }

  assert(false);
}


__device__ void Alive::update_render_array() {
  device_render_cells[cell_id_] = 1;
}


__device__ Candidate::Candidate(int cell_id) : Agent(cell_id) {}


__device__ void Candidate::prepare() {
  int alive_neighbors = num_alive_neighbors();

  if (alive_neighbors == 3) {
    action_ = kActionSpawnAlive;
  } else if (alive_neighbors == 0) {
    action_ = kActionDie;
  }
}


__device__ void Candidate::update() {
  // TODO: Why is this necessary?
  int cid = cell_id_;

  if (action_ == kActionSpawnAlive) {
    cells[cid]->agent_ = device_allocator->make_new<Alive>(cid);
    device_allocator->free<Candidate>(this);
  } else if (action_ == kActionDie) {
    cells[cid]->agent_ = nullptr;
    device_allocator->free<Candidate>(this);
  }
}


__global__ void create_cells() {
  for (int i = threadIdx.x + blockDim.x * blockIdx.x;
       i < SIZE_X*SIZE_Y; i += blockDim.x * gridDim.x) {
    cells[i] = device_allocator->make_new<Cell>();
  }
}


// Must be followed by Alive::update().
__global__ void load_game(int* cell_ids, int num_cells) {
  for (int i = threadIdx.x + blockDim.x * blockIdx.x;
       i < num_cells; i += blockDim.x * gridDim.x) {
    cells[cell_ids[i]]->agent_ =
        device_allocator->make_new<Alive>(cell_ids[i]);
    assert(cells[cell_ids[i]]->agent()->cell_id() == cell_ids[i]);
  }
}


__global__ void initialize_render_arrays() {
  for (int i = threadIdx.x + blockDim.x * blockIdx.x;
       i < SIZE_X*SIZE_Y; i += blockDim.x * gridDim.x) {
    device_render_cells[i] = 0;
  }
}


void render() {
  initialize_render_arrays<<<128, 128>>>();
  gpuErrchk(hipDeviceSynchronize());
  allocator_handle->parallel_do<Alive, &Alive::update_render_array>();

  hipMemcpy(host_render_cells, d_device_render_cells,
             sizeof(char)*dataset.x*dataset.y, hipMemcpyDeviceToHost);
  draw(host_render_cells);
}


void transfer_dataset() {
  int* dev_cell_ids;
  hipMalloc(&dev_cell_ids, sizeof(int)*dataset.num_alive);
  hipMemcpy(dev_cell_ids, dataset.alive_cells, sizeof(int)*dataset.num_alive,
             hipMemcpyHostToDevice);

  printf("Loading on GPU: %i alive cells.\n", dataset.num_alive);
  load_game<<<128, 128>>>(dev_cell_ids, dataset.num_alive);
  gpuErrchk(hipDeviceSynchronize());
  hipFree(dev_cell_ids);
  printf("Done.\n");

  allocator_handle->parallel_do<Alive, &Alive::update>();
}


__device__ int device_checksum;

__device__ void Alive::update_checksum() {
  atomicAdd(&device_checksum, 1);
}


int checksum() {
  int host_checksum = 0;
  hipMemcpyToSymbol(HIP_SYMBOL(device_checksum), &host_checksum, sizeof(int), 0,
                     hipMemcpyHostToDevice);

  allocator_handle->parallel_do<Alive, &Alive::update_checksum>();

  hipMemcpyFromSymbol(&host_checksum, HIP_SYMBOL(device_checksum), sizeof(int), 0,
                       hipMemcpyDeviceToHost);
  return host_checksum;
}


int main(int argc, char** argv) {
  if (argc != 2) {
    printf("Usage: %s filename.pgm\n", argv[0]);
    exit(1);
  } else {
    // Load data set.
    dataset = load_from_file(argv[1]);
  }

  hipMemcpyToSymbol(HIP_SYMBOL(SIZE_X), &dataset.x, sizeof(int), 0,
                     hipMemcpyHostToDevice);
  hipMemcpyToSymbol(HIP_SYMBOL(SIZE_Y), &dataset.y, sizeof(int), 0,
                     hipMemcpyHostToDevice);

  if (OPTION_DRAW) {
    init_renderer();
  }

  AllocatorT::DBG_print_stats();
  
  hipDeviceSetLimit(hipLimitMallocHeapSize, 2*1024U*1024*1024);
  size_t heap_size;
  hipDeviceGetLimit(&heap_size, hipLimitMallocHeapSize);

  // Create new allocator.
  allocator_handle = new AllocatorHandle<AllocatorT>();
  AllocatorT* dev_ptr = allocator_handle->device_pointer();
  hipMemcpyToSymbol(HIP_SYMBOL(device_allocator), &dev_ptr, sizeof(AllocatorT*), 0,
                     hipMemcpyHostToDevice);

  // Allocate memory.
  Cell** host_cells;
  hipMalloc(&host_cells, sizeof(Cell*)*dataset.x*dataset.y);
  hipMemcpyToSymbol(HIP_SYMBOL(cells), &host_cells, sizeof(Cell**), 0,
                     hipMemcpyHostToDevice);

  hipMalloc(&d_device_render_cells, sizeof(char)*dataset.x*dataset.y);
  hipMemcpyToSymbol(HIP_SYMBOL(device_render_cells), &d_device_render_cells,
                     sizeof(char*), 0, hipMemcpyHostToDevice);

  host_render_cells = new char[dataset.x*dataset.y];

  // Initialize cells.
  create_cells<<<128, 128>>>();
  gpuErrchk(hipDeviceSynchronize());

  transfer_dataset();

  auto time_start = std::chrono::system_clock::now();

  // Run simulation.
  for (int i = 0; i < 100; ++i) {
    printf("Iteration %i\n", i);
    allocator_handle->parallel_do<Candidate, &Candidate::prepare>();
    allocator_handle->parallel_do<Alive, &Alive::prepare>();
    allocator_handle->parallel_do<Candidate, &Candidate::update>();
    allocator_handle->parallel_do<Alive, &Alive::update>();

    if (OPTION_DRAW) {
      render();
    }
  }

  if (OPTION_DRAW) {
    close_renderer();
  }

  auto time_end = std::chrono::system_clock::now();
  auto elapsed = time_end - time_start;
  auto millis = std::chrono::duration_cast<std::chrono::milliseconds>(elapsed)
      .count();

  printf("Time: %lu ms\n", millis);

  printf("Checksum: %i\n", checksum());

  delete[] host_render_cells;
  hipFree(host_cells);
  hipFree(d_device_render_cells);

  return 0;
}
