#include "hip/hip_runtime.h"
#include <chrono>
#include <hiprand/hiprand_kernel.h>

#include "barnes_hut.h"
#include "configuration.h"


// Allocator handles.
AllocatorHandle<AllocatorT>* allocator_handle;
__device__ AllocatorT* device_allocator;


// Root of the quad tree.
__DEV__ TreeNode* tree;


template<typename T>
__DEV__ T* pointerCAS(T** addr, T* assumed, T* value) {
  auto* i_addr = reinterpret_cast<unsigned long long int*>(addr);
  auto i_assumed = reinterpret_cast<unsigned long long int>(assumed);
  auto i_value = reinterpret_cast<unsigned long long int>(value);
  return reinterpret_cast<T*>(atomicCAS(i_addr, i_assumed, i_value));
}


__DEV__ NodeBase::NodeBase(TreeNode* parent, float pos_x, float pos_y,
                           float mass)
    : parent_(parent), pos_x_(pos_x), pos_y_(pos_y), mass_(mass) {}


__DEV__ BodyNode::BodyNode(float pos_x, float pos_y, float vel_x, float vel_y,
                           float mass)
    : NodeBase(/*parent=*/ nullptr, pos_x, pos_y, mass),
      vel_x_(vel_x), vel_y_(vel_y) {}


__DEV__ TreeNode::TreeNode(TreeNode* parent, float p1_x, float p1_y,
                           float p2_x, float p2_y)
    : NodeBase(parent, 0.0f, 0.0f, 0.0f),
      p1_x_(p1_x), p1_y_(p1_y), p2_x_(p2_x), p2_y_(p2_y) {
  assert(p1_x < p2_x);
  assert(p1_y < p2_y);
  children_->atomic_write(0, nullptr);
  children_->atomic_write(1, nullptr);
  children_->atomic_write(2, nullptr);
  children_->atomic_write(3, nullptr);

  assert(children_[0] == nullptr);
  assert(children_[1] == nullptr);
  assert(children_[2] == nullptr);
  assert(children_[3] == nullptr);
}


// Set new parent with atomic CAS and retry loop.
__DEV__ void NodeBase::cas_parent_retry(TreeNode* assumed, TreeNode* value) {
  while (parent_.atomic_cas(assumed, value) != assumed) {}
}


__DEV__ float NodeBase::distance_to(NodeBase* other) {
  float dx = other->pos_x() - pos_x_;
  float dy = other->pos_y() - pos_y_;
  return sqrt(dx*dx + dy*dy);
}


__DEV__ void NodeBase::apply_force(BodyNode* body) {
  // Update `body`.
  if (body != this) {
    float dx = body->pos_x() - pos_x_;
    float dy = body->pos_y() - pos_y_;
    float dist = sqrt(dx*dx + dy*dy);
    assert(dist > 0.000000001);  // Should fail only if dist with same body.
    float F = kGravityConstant * mass_ * body->mass()
        / (dist * dist + kDampeningFactor);
    body->add_force(F*dx / dist, F*dy / dist);
  }
}


__DEV__ void BodyNode::compute_force() {
  force_x_ = 0.0f;
  force_y_ = 0.0f;

  // TODO: We may need a while loop here instead of recursion.
  tree->check_apply_force(this);
}


__DEV__ void NodeBase::check_apply_force(BodyNode* body) {
  // TODO: This function should be virtual but we do not have native support
  // for virtual functions in SoaAlloc yet.
  TreeNode* tree_node = this->cast<TreeNode>();
  if (tree_node != nullptr) {
    tree_node->check_apply_force(body);
  } else {
    BodyNode* body_node = this->cast<BodyNode>();
    if (body_node != nullptr) {
      body_node->check_apply_force(body);
    } else {
      assert(false);
    }
  }
}


__DEV__ void TreeNode::check_apply_force(BodyNode* body) {
  if (distance_to(body) <= kDistThreshold) {
    // Too close. Recurse.
    for (int i = 0; i < 4; ++i) {
      if (children_[i] != nullptr) {
        children_[i]->check_apply_force(body);
      }
    }
  } else {
    // Far enough away to use approximation.
    apply_force(body);
  }
}


__DEV__ void BodyNode::check_apply_force(BodyNode* body) {
  apply_force(body);
}


__DEV__ void BodyNode::update() {
  vel_x_ += force_x_*kDt / mass_;
  vel_y_ += force_y_*kDt / mass_;
  pos_x_ += vel_x_*kDt;
  pos_y_ += vel_y_*kDt;

  if (pos_x_ < -1 || pos_x_ > 1) {
    // Undo change in position so that body never leaves boundary tree node.
    pos_x_ -= vel_x_*kDt;
    pos_y_ -= vel_y_*kDt;
    vel_x_ = -vel_x_;
  }

  if (pos_y_ < -1 || pos_y_ > 1) {
    pos_x_ -= vel_x_*kDt;
    pos_y_ -= vel_y_*kDt;
    vel_y_ = -vel_y_;
  }
}


__DEV__ void BodyNode::clear_node() {
  assert(parent_ != nullptr);

  if (!parent_->contains(this)) {
    parent_->remove(this);
    parent_ = nullptr;
  }
}


__DEV__ void TreeNode::remove(NodeBase* body) {
  for (int i = 0; i < 4; ++i) {
    if (children_[i] == body) {
      children_[i] = nullptr;
      return;
    }
  }

  // Node not found.
  assert(false);
}


__DEV__ void BodyNode::add_to_tree() {
  if (parent_ == nullptr) {
    tree->insert(this);
  }
}


__DEV__ int TreeNode::compute_index(BodyNode* body) {
  assert(contains(body));

  // |-----------|
  // |  0  |  1  |
  // |-----|-----|
  // |  2  |  3  |
  // |-----------|

  int c_idx = 0;
  if (body->pos_x() > (p1_x_ + p2_x_) / 2) c_idx = 1;
  if (body->pos_y() > (p1_y_ + p2_y_) / 2) c_idx += 2;
  return c_idx;
}


__DEV__ int TreeNode::child_index(NodeBase* node) {
  int c_idx = - 1;
  for (int i = 0; i < 4; ++i) {
    if (children_.as_volatile()[i] == node) {
      c_idx = i;
      break;
    }
  }
  return c_idx;
}


__DEV__ void TreeNode::insert(BodyNode* body) {
  assert(contains(body));
  TreeNode* current = this;

  bool done = false;
  while (!done) {
    assert(current->contains(body));

    // Check where to insert in this node.
    int c_idx = current->compute_index(body);
    NodeBase* child = current->children_.as_volatile()[c_idx];

    if (child == nullptr) {
      // Empty slot found.
      auto* cas_result = current->children_->atomic_cas(c_idx, nullptr, body);
      if (cas_result == nullptr) {
        // Must set parent with retry loop due to possible race condition.
        // Another thread might already try to insert a TreeNode here.
        body->cas_parent_retry(nullptr, current);

        // Must use while loop condition instead of break from endless loop
        // to avoid deadlock due to branch divergence.
        done = true;
      }  // else: Other thread was faster.
    } else if (child->cast<TreeNode>() != nullptr) {
      current = child->cast<TreeNode>();
    } else {
      BodyNode* other = child->cast<BodyNode>();
      assert(other != nullptr);
      assert(current->contains(other));
      assert(current->compute_index(other) == c_idx);

      // Replace BodyNode with TreeNode.
      float new_p1_x = c_idx == 0 || c_idx == 2
          ? current->p1_x_ : (current->p1_x_ + current->p2_x_) / 2;
      float new_p2_x = c_idx == 0 || c_idx == 2
          ? (current->p1_x_ + current->p2_x_) / 2 : current->p2_x_;
      float new_p1_y = c_idx == 0 || c_idx == 1
          ? current->p1_y_ : (current->p1_y_ + current->p2_y_) / 2;
      float new_p2_y = c_idx == 0 || c_idx == 1
          ? (current->p1_y_ + current->p2_y_) / 2 : current->p2_y_;

      auto* new_node = device_allocator->make_new<TreeNode>(
          /*parent=*/ current, new_p1_x, new_p1_y, new_p2_x, new_p2_y);
      assert(new_node->contains(other));
      assert(new_node->contains(body));

      // Insert other into new node.
      // This could be a volatile write with threadfence. But atomic is safer.
      int other_idx = new_node->compute_index(other);
#ifndef NDEBUG
      assert(new_node->children_->atomic_cas(other_idx, nullptr, other)
             == nullptr);
#else
      new_node->children_->atomic_write(other_idx, other);
#endif  // NDEBUG

      // Try to install this node.
      if (current->children_->atomic_cas(c_idx, other, new_node) == other) {
        other->cas_parent_retry(current, new_node);

        // Now insert body.
        current = new_node;
      } else {
        device_allocator->free(new_node);
      }
    }
  }

#ifndef NDEBUG
  body->sanity_check();
#endif  // NDEBUG
}


__DEV__ bool TreeNode::remove_child(int c_idx, TreeNode* node) {
  NodeBase* before = children_->atomic_cas(c_idx, node, nullptr);

#ifndef NDEBUG
  assert(before == nullptr || before == node);
#endif  // NDEBUG

  return before == node;
}


__DEV__ void TreeNode::collapse_tree() {
  // Collapse bottom-up.
  // Leaf = Only BodyNode objects as children. Or no children at all.

  if (is_leaf()) {
    TreeNode* current = this;

    while (current != tree) {
      TreeNode* parent = current->parent_.as_volatile();
      assert(parent != nullptr);

      int num_children = 0;
      NodeBase* single_child = nullptr;

      for (int i = 0; i < 4; ++i) {
        // TODO: There could be cases where we do not see a concurrent delete
        // due to missing threadfence.
        // Dangerous: Multiple threads may be deleting stuff at the same time.
        auto* child = current->children_.as_volatile()[i];
        if (child != nullptr) {
          ++num_children;
          single_child = child;
        }
      }

      if (num_children < 2) {
        // Find index of current node in parent.
        // TODO: Consider using compute_index instead.
        int c_idx = parent_->child_index(current);

        if (c_idx != -1) {
          if (num_children == 0) {
            // Node is empty. Remove.
            if (parent->remove_child(c_idx, current)) {
              current = parent;
              device_allocator->free(current);
            } else {
              // Another thread already remove this node.
              break;
            }
          } else if (num_children == 1) {
#ifndef NDEBUG
            assert(single_child != nullptr);
            BodyNode* child_body = single_child->cast<BodyNode>();
            if (child_body != nullptr) {
              assert(current->contains(child_body));
              assert(parent->contains(child_body));
              assert(parent->compute_index(child_body) == c_idx);
            }
#endif  // NDEBUG

            // Node has only one child. Merge with parent.
            NodeBase* before = parent->children_->atomic_cas(
                c_idx, current, single_child);

            if (before == current) {
              assert(single_child->parent() == current);
              // TODO: Use pointerCAS here?
              single_child->set_parent(parent);
              device_allocator->free(current);
              current = parent;
            } else {
              // Another thread already performed a merge or removed the node.
              break;
            }
          }
        } else {
          // Node not found in parent. Other thread modified node.
          break;
        }
      } else {
         // Retain node.
        break;
      }
    }
  }
}


__DEV__ bool TreeNode::is_leaf() {
  // A node is a leaf if it has at least one BodyNode child and no TreeNode
  // child.
  bool has_body_node = false;
  for (int i = 0; i < 4; ++i) {
    if (children_[i]->cast<TreeNode>() != nullptr) {
      return false;
    } else if (children_[i]->cast<BodyNode>() != nullptr) {
      has_body_node = true;
    }
  }

  return has_body_node;
}


__DEV__ bool TreeNode::contains(BodyNode* body) {
  float x = body->pos_x();
  float y = body->pos_y();
  return x >= p1_x_ && x < p2_x_ && y >= p1_y_ && y < p2_y_;
}


__DEV__ void TreeNode::initialize_frontier() {
  frontier_ = is_leaf();
  next_frontier_ = false;
  visited_ = false;
}


__DEV__ void TreeNode::update_frontier() {
  frontier_ = next_frontier_;
  next_frontier_ = false;
}


__DEV__ void TreeNode::bfs_step() {
  if (frontier_) {
    visited_ = true;
    frontier_ = false;

    // Update pos_x and pos_y: gravitational center
    float total_mass = 0.0f;
    float sum_pos_x = 0.0f;
    float sum_pos_y = 0.0f;

    for (int i = 0; i < 4; ++i) {
      if (children_[i] != nullptr) {
        total_mass += children_[i]->mass();
        sum_pos_x += children_[i]->mass()*children_[i]->pos_x();
        sum_pos_y += children_[i]->mass()*children_[i]->pos_y();

#ifndef NDEBUG
        BodyNode* body_node = children_[i]->cast<BodyNode>();
        if (body_node != nullptr) {
          // Ensure that BodyNodes are properly initialized.
          assert(body_node->mass() > 0.000000001);
        }
#endif  // NDEBUG
      }
    }

    assert(total_mass > 0.000000001);  // Should fail only if empty node.
    pos_x_ = sum_pos_x/total_mass;
    pos_y_ = sum_pos_y/total_mass;
    mass_ = total_mass;

    // Add parent to frontier.
    if (parent_ != nullptr) {
      parent_->next_frontier_ = true;
    } else {
      assert(this == tree);
    }
  }
}


__DEV__ void TreeNode::remove_unvisited() {
  if (!visited_) {
    // Remove this node.
    assert(parent_ != nullptr);
    parent_->remove(this);
    device_allocator->free(this);
  }
}


__DEV__ void BodyNode::sanity_check() {
  // BodyNode is part of the tree.
  assert(parent_ != nullptr);

  // Node is properly registered in the parent.
  bool found = false;
  for (int i = 0; i < 4; ++i) {
    if (parent_->child(i) == this) {
      found = true;
      break;
    }
  }
  assert(found);
}


__DEV__ void TreeNode::sanity_check() {
  // BodyNode is part of the tree.
  if (this != tree) {
    assert(parent_ != nullptr);

    // Node is properly registered in the parent.
    bool found = false;
    for (int i = 0; i < 4; ++i) {
      if (parent_->child(i) == this) {
        found = true;
        break;
      }
    }
    assert(found);
  } else {
    assert(parent_ != tree);
  }
}


void bfs() {
  // BFS steps to update tree.
  allocator_handle->parallel_do<TreeNode, &TreeNode::initialize_frontier>();
  for (int i = 0; i < 100; ++i) {
    allocator_handle->parallel_do<TreeNode, &TreeNode::bfs_step>();
    allocator_handle->parallel_do<TreeNode, &TreeNode::update_frontier>();
  }

  allocator_handle->parallel_do<TreeNode, &TreeNode::remove_unvisited>();
}


void step() {
#ifndef NDEBUG
  printf("A\n");
  allocator_handle->parallel_do<BodyNode, &BodyNode::sanity_check>();
  allocator_handle->parallel_do<TreeNode, &TreeNode::sanity_check>();
  printf("A done\n");
#endif  // NDEBUG

  allocator_handle->parallel_do<BodyNode, &BodyNode::compute_force>();
  allocator_handle->parallel_do<BodyNode, &BodyNode::update>();
  allocator_handle->parallel_do<BodyNode, &BodyNode::clear_node>();
  allocator_handle->parallel_do<BodyNode, &BodyNode::add_to_tree>();
  //allocator_handle->parallel_do<TreeNode, &TreeNode::collapse_tree>();

#ifndef NDEBUG
  printf("B\n");
  allocator_handle->parallel_do<BodyNode, &BodyNode::sanity_check>();
  allocator_handle->parallel_do<TreeNode, &TreeNode::sanity_check>();
  printf("B done\n");
#endif  // NDEBUG


  bfs();
}


__global__ void kernel_init_tree() {
  tree = device_allocator->make_new<TreeNode>(
      /*parent=*/ nullptr,
      /*p1_x=*/ -1.0f,
      /*p1_y=*/ -1.0f,
      /*p2_x=*/ 1.0f,
      /*p2_y=*/ 1.0f);
}


__global__ void kernel_init_bodies() {
  int tid = threadIdx.x + blockDim.x * blockIdx.x;
  hiprandState rand_state;
  hiprand_init(kSeed, tid, 0, &rand_state);

  for (int i = tid; i < kNumBodies; i += blockDim.x * gridDim.x) {
    device_allocator->make_new<BodyNode>(
        /*pos_x=*/ 2 * hiprand_uniform(&rand_state) - 1,
        /*pos_y=*/ 2 * hiprand_uniform(&rand_state) - 1,
        /*vel_x=*/ (hiprand_uniform(&rand_state) - 0.5) / 1000,
        /*vel_y=*/ (hiprand_uniform(&rand_state) - 0.5) / 1000,
        /*mass=*/ (hiprand_uniform(&rand_state)/2 + 0.5) * kMaxMass);
  }
}


__device__ double device_checksum;
__DEV__ void BodyNode::add_checksum() {
  device_checksum += pos_x_ + pos_y_*2 + vel_x_*3 + vel_y_*4;
}


__global__ void kernel_compute_checksum() {
  device_checksum = 0.0f;
  device_allocator->template device_do<BodyNode>(&BodyNode::add_checksum);
}


void initialize_simulation() {
  kernel_init_tree<<<1, 1>>>();
  gpuErrchk(hipDeviceSynchronize());

  kernel_init_bodies<<<128, 128>>>();
  gpuErrchk(hipDeviceSynchronize());

  allocator_handle->parallel_do<BodyNode, &BodyNode::add_to_tree>();
  bfs();
}


int main(int /*argc*/, char** /*argv*/) {
  // Create new allocator.
  allocator_handle = new AllocatorHandle<AllocatorT>();
  AllocatorT* dev_ptr = allocator_handle->device_pointer();
  hipMemcpyToSymbol(HIP_SYMBOL(device_allocator), &dev_ptr, sizeof(AllocatorT*), 0,
                     hipMemcpyHostToDevice);

  initialize_simulation();

  for (int i = 0; i < kIterations; ++i) {
    printf("STEP: %i\n", i);
    step();
  }

  kernel_compute_checksum<<<1, 1>>>();
  gpuErrchk(hipDeviceSynchronize());

  double checksum;
  hipMemcpyFromSymbol(&checksum, HIP_SYMBOL(device_checksum), sizeof(device_checksum), 0,
                       hipMemcpyDeviceToHost);
  printf("Checksum: %f\n", checksum);
}
